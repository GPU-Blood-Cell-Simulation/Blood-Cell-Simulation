#include "hip/hip_runtime.h"
#include "utilities.cuh"

__host__ __device__ float3 operator*(float a, float3 v)
{
	return make_float3(a * v.x, a * v.y, a * v.z);
}

__host__ __device__ float3 operator*(float3 a, float3 b)
{
	return make_float3(a.x * b.x, a.y * b.y, a.z * b.z);
}

__host__ __device__ float3 operator+(float3 a, float3 b)
{
	return make_float3(a.x + b.x, a.y + b.y, a.z + b.z);
}

__host__ __device__ float3 operator-(float3 a, float3 b)
{
	return make_float3(a.x - b.x, a.y - b.y, a.z - b.z);
}

__host__ __device__ float3 operator/(float3 v, float a)
{
	return make_float3(v.x / a, v.y / a, v.z / a);
}

__host__ __device__ float dot(float3 a, float3 b)
{
	return a.x * b.x + a.y * b.y + a.z * b.z;
}

__host__ __device__ float3 cross(float3 u, float3 v)
{
	return make_float3(u.y * v.z - u.z * v.y,
		u.z * v.x - u.x * v.z,
		u.x * v.y - u.y * v.x);
}

__host__ __device__ float length(float3 v)
{
	return sqrtf(v.x * v.x + v.y * v.y + v.z * v.z);
}

__host__ __device__ float3 normalize(float3 v) // versor
{
	return v / sqrtf(dot(v, v));
}