#include "cuda_vec3.cuh"

#include "cuda_handle_error.cuh"

cudaVec3::cudaVec3(int n) : size(n)
{
	HANDLE_ERROR(hipMalloc((void**)&x, n * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&y, n * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&z, n * sizeof(float)));
}

cudaVec3::cudaVec3(const cudaVec3& other) : isCopy(true), x(other.x), y(other.y), z(other.z), size(other.size) {}

cudaVec3::~cudaVec3()
{
	if (!isCopy)
	{
		HANDLE_ERROR(hipFree(x));
		HANDLE_ERROR(hipFree(y));
		HANDLE_ERROR(hipFree(z));
	}
}