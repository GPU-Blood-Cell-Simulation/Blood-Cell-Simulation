#include "hip/hip_runtime.h"
#include "uniform_grid.cuh"
#include "../defines.hpp"
#include "../utilities/cuda_handle_error.cuh"

#include <cstdio>
#include <cstdlib>

#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>

#include "hip/hip_runtime.h"
#include ""

#pragma region Helper kernels

__device__ unsigned int calculateIdForCell(float x, float y, float z)
{
	return
		static_cast<unsigned int>(z / cellDepth) * static_cast<unsigned int>(width / cellWidth) * static_cast<unsigned int>(height / cellHeight) +
		static_cast<unsigned int>(y / cellHeight) * static_cast<unsigned int>(width / cellWidth) +
		static_cast<unsigned int>(x / cellWidth);
}

__global__ void calculateCellIdKernel(const float* positionX, const float* positionY, const float* positionZ,
	unsigned int* cellIds, unsigned int* particleIds, const unsigned int particleCount)
{
	unsigned int particleId = blockIdx.x * blockDim.x + threadIdx.x;
	if (particleId >= particleCount)
		return;

	/*unsigned int cellId =
		static_cast<unsigned int>(positionZ[particleId] / cellDepth) * static_cast<unsigned int>(width / cellWidth) * static_cast<unsigned int>(height / cellHeight) +
		static_cast<unsigned int>(positionY[particleId] / cellHeight) * static_cast<unsigned int>(width / cellWidth) +
		static_cast<unsigned int>(positionX[particleId] / cellWidth);*/
	unsigned int cellId = calculateIdForCell(positionX[particleId], positionY[particleId], positionZ[particleId]);
	// Debug
	/*if (cellId >= 9261)
		printf("Error, cellId: %d\n", cellId);*/

	//printf("id: %d, cellId: %d\n", particleId, cellId);

	particleIds[particleId] = particleId;
	cellIds[particleId] = cellId;

}

__global__ void calculateStartAndEndOfCellKernel(const float* positionX, const float* positionY, const float* positionZ,
	const unsigned int* cellIds, const unsigned int* particleIds,
	unsigned int* cellStarts, unsigned int* cellEnds, unsigned int particleCount)
{
	unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id >= particleCount)
		return;

	unsigned int currentCellId = cellIds[id];

	// Check if the previous cell id was different - it would mean we found the start of a cell
	if (id > 0 && currentCellId != cellIds[id - 1])
	{
		cellStarts[currentCellId] = id;
	}

	// Check if the next cell id was different - it would mean we found the end of a cell
	if (id < particleCount - 1 && currentCellId != cellIds[id + 1])
	{
		cellEnds[currentCellId] = id;
	}

	if (id == 0)
	{
		cellStarts[cellIds[0]] = 0;
	}
	if (id == particleCount - 1)
	{
		cellStarts[cellIds[particleCount - 1]] = particleCount - 1;
	}
}

#pragma endregion

// Allocate GPU buffers for the index buffers
UniformGrid::UniformGrid(const unsigned int particleCount)
{
	cellAmount = width / cellWidth * height / cellHeight * depth / cellDepth;
	HANDLE_ERROR(hipMalloc((void**)&gridCellIds, particleCount * sizeof(unsigned int)));
	HANDLE_ERROR(hipMalloc((void**)&particleIds, particleCount * sizeof(unsigned int)));

	HANDLE_ERROR(hipMalloc((void**)&gridCellStarts, cellAmount * sizeof(unsigned int)));
	HANDLE_ERROR(hipMalloc((void**)&gridCellEnds, cellAmount * sizeof(unsigned int)));
}

UniformGrid::UniformGrid(const UniformGrid& other) : isCopy(true), gridCellIds(other.gridCellIds), particleIds(other.particleIds),
	gridCellStarts(other.gridCellStarts), gridCellEnds(other.gridCellEnds) 
{}

UniformGrid::~UniformGrid()
{
	if (!isCopy)
	{
		HANDLE_ERROR(hipFree(gridCellIds));
		HANDLE_ERROR(hipFree(particleIds));
		HANDLE_ERROR(hipFree(gridCellStarts));
		HANDLE_ERROR(hipFree(gridCellEnds));
	}
	
}

void UniformGrid::calculateGrid(const float* positionX, const float* positionY, const float* positionZ, unsigned int objectCount)
{
	// Calculate launch parameters

	const int threadsPerBlock = objectCount > 1024 ? 1024 : objectCount;
	const int blocks = (objectCount + threadsPerBlock - 1) / threadsPerBlock;

	// 1. Calculate cell id for every particle and store as pair (cell id, particle id) in two buffers
	calculateCellIdKernel << <blocks, threadsPerBlock >> >
		(positionX, positionY, positionZ, gridCellIds, particleIds, objectCount);

	// 2. Sort particle ids by cell id

	thrust::device_ptr<unsigned int> keys = thrust::device_pointer_cast<unsigned int>(gridCellIds);
	thrust::device_ptr<unsigned int> values = thrust::device_pointer_cast<unsigned int>(particleIds);

	thrust::stable_sort_by_key(keys, keys + objectCount, values);

	// 3. Find the start and end of every cell

	calculateStartAndEndOfCellKernel << <blocks, threadsPerBlock >> >
		(positionX, positionY, positionZ, gridCellIds, particleIds, gridCellStarts, gridCellEnds, objectCount);
}

__device__ unsigned int UniformGrid::calculateCellId(float3 position)
{
	return calculateIdForCell(position.x, position.y, position.z);
}