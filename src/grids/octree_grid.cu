#include "hip/hip_runtime.h"
#include "octree_grid.cuh"
#include "grid_helpers.cuh"
#include "../defines.hpp"
#include "../utilities/cuda_handle_error.cuh"
#include "../utilities/math.cuh"
#include "math_functions.h"

__device__ unsigned int partEveryByteByTwo(unsigned int n)
{
	n = (n ^ (n << 16)) & 0xff0000ff;
	n = (n ^ (n << 8)) & 0x0300f00f;
	n = (n ^ (n << 4)) & 0x030c30c3;
	n = (n ^ (n << 2)) & 0x09249249;
	return n;
}

__device__ unsigned int calculateMortonCodeIdForCell(unsigned int xId, unsigned int yId, unsigned int zId)
{
	return (partEveryByteByTwo(zId) << 2) | (partEveryByteByTwo(yId) << 1) | partEveryByteByTwo(xId);
}

__global__ void calculateCellIdKernel(const float* positionX, const float* positionY, const float* positionZ,
	unsigned int* cellIds, unsigned int* particleIds, const unsigned int particleCount,
	unsigned int cellWidth, unsigned int cellHeight, unsigned int cellDepth)
{
	unsigned int particleId = blockIdx.x * blockDim.x + threadIdx.x;
	if (particleId >= particleCount)
		return;

	unsigned int cellId = calculateMortonCodeIdForCell(unsigned int(positionX[particleId] / cellWidth),
		unsigned int(positionY[particleId] / cellHeight), unsigned int(positionZ[particleId] / cellDepth));

	particleIds[particleId] = particleId;
	cellIds[particleId] = cellId;
}

__device__ unsigned int calculateCellIdFromMorton(unsigned int mortonCode, unsigned int levels)
{
	unsigned int mask = 8 << 3 * levels;
	unsigned int realId = 0;

#pragma unroll
	for (int i = 3 * levels; i >= 0; i -= 3) {
		realId += (mask & mortonCode) >> i;
		realId *= 8;
	}
	return realId;
}

__global__ void calculateCellStarts(const unsigned int* cellIds, const unsigned int* particleIds,
	unsigned int* treeData, unsigned int cellCount, unsigned int levels)
{
	unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id >= cellCount)
		return;

	unsigned int currentCellId = cellIds[id];

	if (id == 0 || cellIds[id - 1] >> 3 < currentCellId >> 3)
	{
		levels -= 2;
		unsigned int realId = calculateCellIdFromMorton(currentCellId, levels);
		unsigned int index = realId - (unsigned int)(pow(8, levels) - 1) / 7;
		treeData[index] = id;
	}
}

void createOctreeGridData(const float* positionX, const float* positionY, const float* positionZ, unsigned int* gridCellIds,
	unsigned int* particleIds, float cellWidth, float cellHeight, float cellDepth, /*unsigned int* gridCelLStarts,
	unsigned int* gridCellEnds,*/ unsigned int objectCount)
{
	// Calculate launch parameters

	const int threadsPerBlock = objectCount > 1024 ? 1024 : objectCount;
	const int blocks = (objectCount + threadsPerBlock - 1) / threadsPerBlock;

	// 1. Calculate cell id for every particle and store as pair (cell id, particle id) in two buffers
	calculateCellIdKernel << <blocks, threadsPerBlock >> >
		(positionX, positionY, positionZ, gridCellIds, particleIds, objectCount, cellWidth, cellHeight, cellDepth);

	// 2. Sort particle ids by cell id

	thrust::device_ptr<unsigned int> keys = thrust::device_pointer_cast<unsigned int>(gridCellIds);
	thrust::device_ptr<unsigned int> values = thrust::device_pointer_cast<unsigned int>(particleIds);

	thrust::stable_sort_by_key(keys, keys + objectCount, values);

	// 3. Find the start and end of every cell
	calculateCellStarts(gridCellIds, particleIds, treeData, objectCount, levels);
	/*gridHelpers::calculateStartAndEndOfCellKernel << <blocks, threadsPerBlock >> >
		(positionX, positionY, positionZ, gridCellIds, particleIds, gridCellStarts, gridCellEnds, objectCount);*/
}




__global__ void calculateTreeLeafsCells(const unsigned int* cellIds, const unsigned int objectCount,
	const unsigned int cellCountX, const unsigned int cellCountY, const unsigned int cellCountZ,
	const unsigned int cellWidth, const unsigned int cellHeight, const unsigned int cellDepth, const unsigned int divisionCount
	, const float width, const float height, const float depth, unsigned char* masks)
{
	unsigned int objectId = blockIdx.x * blockDim.x + threadIdx.x;
	if (objectId >= objectCount)
		return;

	unsigned int cellMortonCodeId = cellIds[objectId];
	unsigned int currentCellAbsoluteId = cellMortonCodeId + (pow(8, divisionCount) - 1) / 7;
	unsigned int counter = 0;


#pragma unroll
	while (counter++ < divisionCount - 1) {
		unsigned int parentId = currentCellAbsoluteId >> 3;
		unsigned char childFillMask = 1 << (currentCellAbsoluteId & MORTON_POSITION_MASK);

		if (!(masks[parentId] & childFillMask))
			atomicOr_system(masks + parentId, childFillMask);
			masks[parentId] |= childFillMask;
		
		currentCellAbsoluteId = parentId;
	}

}


OctreeGrid::OctreeGrid(const unsigned int objectCount, const unsigned int levels)
{
	this->objectsCount = objectCount;
	this->levels = levels;

	cellWidth = width / pow(2, levels);
	cellHeight = height / pow(2, levels);
	cellDepth = depth / pow(2, levels);


	cellCountX = static_cast<unsigned int>(width / cellWidth);
	cellCountY = static_cast<unsigned int>(height / cellHeight);
	cellCountZ = static_cast<unsigned int>(depth / cellDepth);

	cellAmount = width / cellWidth * height / cellHeight * depth / cellDepth;
	HANDLE_ERROR(hipMalloc((void**)&gridCellIds, objectsCount * sizeof(unsigned int)));
	HANDLE_ERROR(hipMalloc((void**)&particleIds, objectsCount * sizeof(unsigned int)));

	//HANDLE_ERROR(hipMalloc((void**)&gridCellStarts, cellAmount * sizeof(unsigned int)));
	//HANDLE_ERROR(hipMalloc((void**)&gridCellEnds, cellAmount * sizeof(unsigned int)));

	treeNodesCount = (pow(8, levels) - 1) / 7;
	printf("Octree nodes count: %d\n", treeNodesCount);
	HANDLE_ERROR(hipMalloc((void**)&masks, treeNodesCount * sizeof(unsigned char)));
	
	lastNonLeafLayerCount = pow(8, levels - 2);
	HANDLE_ERROR(hipMalloc((void**)&treeData, lastNonLeafLayerCount * sizeof(unsigned int)));
}

void OctreeGrid::calculateGrid(const float* positionX, const float* positionY, const float* positionZ, unsigned int particleCount)
{
	createOctreeGridData(positionX, positionY, positionZ, gridCellIds, particleIds, cellWidth,
		cellHeight, cellDepth, /*gridCellStarts, gridCellEnds, */particleCount);


}