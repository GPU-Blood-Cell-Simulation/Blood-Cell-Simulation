#include "hip/hip_runtime.h"
#include "octree_grid.cuh"
#include "grid_helpers.cuh"
#include "../defines.hpp"
#include "../utilities/cuda_handle_error.cuh"
#include "../utilities/math.cuh"
#include "math_functions.h"

#define EXPONENTIAL_MASK 0x7f800000
#define EXPONENTIAL_OFFSET 23
#define MANTIS_MASK 0x007fffff
#define MANTIS_OFFSET_LEFTSHIFTED 9

#define MANTIS_OR_MASK 0x00800000


#define MORTON_POSITION_MASK 0x07


struct positive_float_structure {

	unsigned int mantis;
	unsigned char exponent;
	
	positive_float_structure(float value) {
		unsigned int valueCasted = *(int*)&value;
		exponent = (valueCasted & EXPONENTIAL_MASK) >> EXPONENTIAL_OFFSET;
		mantis = valueCasted << MANTIS_OFFSET_LEFTSHIFTED;
	}
};

__device__ unsigned int partEveryByteByTwo(unsigned int n)
{
	n = (n ^ (n << 16)) & 0xff0000ff;
	n = (n ^ (n << 8)) & 0x0300f00f;
	n = (n ^ (n << 4)) & 0x030c30c3;
	n = (n ^ (n << 2)) & 0x09249249;
	return n;
}

__device__ unsigned int calculateMortonCodeIdForCell(unsigned int xId, unsigned int yId, unsigned int zId)
{
	return (partEveryByteByTwo(zId) << 2) | (partEveryByteByTwo(yId) << 1) | partEveryByteByTwo(xId);
}

__global__ void calculateCellIdKernel(const float* positionX, const float* positionY, const float* positionZ,
	unsigned int* cellIds, unsigned int* particleIds, const unsigned int particleCount,
	unsigned int cellWidth, unsigned int cellHeight, unsigned int cellDepth)
{
	unsigned int particleId = blockIdx.x * blockDim.x + threadIdx.x;
	if (particleId >= particleCount)
		return;

	unsigned int cellId = calculateMortonCodeIdForCell(unsigned int(positionX[particleId] / cellWidth),
		unsigned int(positionY[particleId] / cellHeight), unsigned int(positionZ[particleId] / cellDepth));

	particleIds[particleId] = particleId;
	cellIds[particleId] = cellId;
}

__global__ void calculateCellStarts(const unsigned int* cellIds, const unsigned int* particleIds,
	unsigned int* treeData, unsigned int cellCount, unsigned int levels)
{
	unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id >= cellCount)
		return;

	unsigned int currentCellId = cellIds[id];

	if (id == 0 || cellIds[id - 1] >> 3 < currentCellId >> 3)
	{
		levels -= 2;
		unsigned int mask = 8 << 3 * levels;
		unsigned int realId = 0;

#pragma unroll
		for (int i = 3 * levels; i >= 0; i -= 3) {
			realId += (mask & currentCellId) >> i;
			realId *= 8;
		}

		unsigned int index = realId - (unsigned int)(pow(8, levels) - 1) / 7;
		treeData[index] = id;
	}
}

void createOctreeGridData(const float* positionX, const float* positionY, const float* positionZ, unsigned int* gridCellIds,
	unsigned int* particleIds, float cellWidth, float cellHeight, float cellDepth, /*unsigned int* gridCelLStarts,
	unsigned int* gridCellEnds,*/ unsigned int objectCount)
{
	// Calculate launch parameters

	const int threadsPerBlock = objectCount > 1024 ? 1024 : objectCount;
	const int blocks = (objectCount + threadsPerBlock - 1) / threadsPerBlock;

	// 1. Calculate cell id for every particle and store as pair (cell id, particle id) in two buffers
	calculateCellIdKernel << <blocks, threadsPerBlock >> >
		(positionX, positionY, positionZ, gridCellIds, particleIds, objectCount, cellWidth, cellHeight, cellDepth);

	// 2. Sort particle ids by cell id

	thrust::device_ptr<unsigned int> keys = thrust::device_pointer_cast<unsigned int>(gridCellIds);
	thrust::device_ptr<unsigned int> values = thrust::device_pointer_cast<unsigned int>(particleIds);

	thrust::stable_sort_by_key(keys, keys + objectCount, values);

	// 3. Find the start and end of every cell
	calculateCellStarts(gridCellIds, particleIds, treeData, objectCount, levels);
	/*gridHelpers::calculateStartAndEndOfCellKernel << <blocks, threadsPerBlock >> >
		(positionX, positionY, positionZ, gridCellIds, particleIds, gridCellStarts, gridCellEnds, objectCount);*/
}




__global__ void calculateTreeLeafsCells(const unsigned int* cellIds, const unsigned int objectCount,
	const unsigned int cellCountX, const unsigned int cellCountY, const unsigned int cellCountZ,
	const unsigned int cellWidth, const unsigned int cellHeight, const unsigned int cellDepth, const unsigned int divisionCount
	, const float width, const float height, const float depth, unsigned char* masks)
{
	unsigned int objectId = blockIdx.x * blockDim.x + threadIdx.x;
	if (objectId >= objectCount)
		return;

	unsigned int cellMortonCodeId = cellIds[objectId];
	unsigned int currentCellAbsoluteId = cellMortonCodeId + (pow(8, divisionCount) - 1) / 7;
	unsigned int counter = 0;


#pragma unroll
	while (counter++ < divisionCount - 1) {
		unsigned int parentId = currentCellAbsoluteId >> 3;
		unsigned char childFillMask = 1 << (currentCellAbsoluteId & MORTON_POSITION_MASK);

		if (!(masks[parentId] & childFillMask))
			atomicOr_system(masks + parentId, childFillMask);
			masks[parentId] |= childFillMask;
		
		currentCellAbsoluteId = parentId;
	}

}

// transformed
__device__ unsigned int calculateCellForPosition(float3 position, float3 cellCenter)
{
	return ((cellCenter.z < position.z) << 2) | ((cellCenter.y < position.y) << 1) | (cellCenter.x < position.x);
}


__device__ float3 calculateChildCenter(float3 center, unsigned int childId, float3 childCellDimension)
{
	return center + make_float3(((childId & 1) ? childCellDimension.x : -childCellDimension.x),
		((childId & 2) ? childCellDimension.y : -childCellDimension.y), ((childId & 4) ? childCellDimension.z : -childCellDimension.z));
}

__device__ float3 calculateParentCenter(float3 center, unsigned int childId)
{
	return center + make_float3((!(childId & 1) ? center.x / 2 : -center.x / 2),
		(!(childId & 2) ? center.y / 2 : -center.y / 2), (!(childId & 4) ? center.z / 2 : -center.z / 2));
}

__device__ float3 calculateRayTValue(float3 origin, float3 inversedDirection, float3 argument)
{
	return (argument - origin) * inversedDirection;
}

__device__ float3 calculateLeafCellFromMorton(float3 cellDimension, float3 bounding, unsigned int mortonCode, unsigned int level) {
	
	float3 leafCell = make_float3(0, 0, 0);
#pragma unroll
	for (int i = 0; i < level - 1; i++) {
		unsigned int mask = mortonCode & 8;
		bounding = bounding / 2;
		if (mask & 1) {
			leafCell.x += bounding.x;
		}
		if (mask & 2) {
			leafCell.y += bounding.y;
		}
		if (mask & 4) {
			leafCell.z += bounding.z;
		}
		mortonCode >> 3;
	}
	return leafCell;
}

__device__ float3 calculateNeighbourLeafPos(float3 pos, float3 direction, float3 childCellSize, unsigned char bitChange)
{
	float3 newPos = pos;
	if (bitChange > 1) {
		if (direction.z < 0)
			newPos.z -= childCellSize.z;
		else
			newPos.z += childCellSize.z;
	}
	else if (bitChange) {
		if (direction.y < 0)
			newPos.y -= childCellSize.y;
		else
			newPos.y += childCellSize.y;
	}
	else {
		if (direction.x < 0)
			newPos.x -= childCellSize.x;
		else
			newPos.x += childCellSize.x;
	}
	return newPos;

}

__device__ void traverseGrid(float3 origin, float3 direction, float tmax, unsigned char* masks, unsigned int* treeData, const unsigned int maxLevel)
{
	// necessary parameters
	const float3 bounding = make_float3(width, height, depth);
	const unsigned int s_max = maxLevel;
	const unsigned int leafShift = (ldexp(1, 3 * (maxLevel - 1)) - 1) / 7; // (8^(maxL - 1) - 1)/7
	const float3 relativeOrigin = origin / bounding;
	const float3 directionSigns = make_float3(!(direction.x < 0), !(direction.y < 0), !(direction.z < 0)); // 1 plus or zero, 0 minus
	const float3 inversedDirection = make_float3(1.0f / direction.x, 1.0f / direction.y, 1.0f / direction.z);

	// values
	unsigned int parentId = 0; // root
	float3 pos = { 0.5f, 0.5f, 0.5f }; // initial pos is center of root
	unsigned int scale = s_max - 1;
	unsigned int childId = calculateCellForPosition(relativeOrigin, pos);
	unsigned int realChildId = 8 * parentId + childId + 1;
	float3 childCellSize = 0.5f*bounding;
	float3 tBegin = relativeOrigin;
	float3 tEnd = relativeOrigin;

	while (true) {

		// traversing down the stack
		if (scale > 0) {
			childCellSize = 0.5f*childCellSize;
			parentId = realChildId;
			pos = calculateChildCenter(pos, childId, childCellSize);
			childId = calculateCellForPosition(relativeOrigin, pos);
			unsigned int realChildId = 8 * parentId + childId + 1;
			
			if (!(masks[realChildId] & (1 << childId))) // empty cell
				break;

			//parentCellCenter = tempCenter; // maybe do not need tempCenter ??? 
			scale--;
			//scale = scale / 2;
			continue;
		}

		// compute intersections in current cell
		// TODO

		// calculate neighbour cell
		unsigned int leafMortonCode = treeData[realChildId - leafShift];

		//float3 cellBegining = calculateLeafCellFromMorton(scale, bounding, leafMortonCode, currentLevel);
		float3 cellBeginning = pos - make_float3(fmodf(pos.x, childCellSize.x),
			fmodf(pos.y, childCellSize.y), fmodf(pos.z, childCellSize.z));

		tBegin = tEnd;

		// maybe ifs instead of directionSigns ???
		tEnd = calculateRayTValue(relativeOrigin, inversedDirection, cellBeginning + directionSigns*childCellSize);
		float tMax = vmin(tEnd);

		bool changeParent = false;

		unsigned char bitChange = 0;
		// bit changing && should be + && is minus
		if (!(tMax > tEnd.x) && (childId & 1) && direction.x < 0) {
			changeParent = true;
		}
		else if (!(tMax > tEnd.x)) {
			bitChange = 1;
			if ((childId & 1) && direction.y < 0) {
				changeParent = true;
			}
		}
		else if (!(tMax > tEnd.x)) {
			bitChange = 2;
			if ((childId & 1) && direction.z < 0) {
				changeParent = true;
			}
		}
		
		if (changeParent) {
			// calculate new pos
			float3 newPos = calculateNeighbourLeafPos(pos, direction, childCellSize, bitChange);

			positive_float_structure posX(pos.x), posY(pos.y), posZ(pos.z);
			positive_float_structure newPosX(newPos.x), newPosY(newPos.y), newPosZ(newPos.z);

			unsigned char minBinaryPlacesPos = 127 - max(posX.exponent, max(posY.exponent, posZ.exponent));
			unsigned char minBinaryPlacesNewPos = 127 - max(newPosX.exponent, max(newPosY.exponent, newPosZ.exponent));

			if (minBinaryPlacesPos > minBinaryPlacesNewPos) {
				scale = s_max - minBinaryPlacesNewPos;
				childCellSize = ldexp(1, scale - s_max) * bounding;
				// childId =
				// parentId = 
			}
			else if (minBinaryPlacesPos < minBinaryPlacesNewPos) {

			}
			else { // the same, check mantisa

			}
		}
		else {

			// calculate new childId
			childId ^= 1 << bitChange;
			unsigned int realChildId = 8 * parentId + childId + 1;

			if (!(masks[realChildId] & (1 << childId))) // empty cell
				break;

			// calculate new pos
			pos = calculateNeighbourLeafPos(pos, direction, childCellSize, bitChange);
		}
	}
}

OctreeGrid::OctreeGrid(const unsigned int objectCount, const unsigned int levels)
{
	this->objectsCount = objectCount;
	this->levels = levels;

	cellWidth = width / pow(2, levels);
	cellHeight = height / pow(2, levels);
	cellDepth = depth / pow(2, levels);


	cellCountX = static_cast<unsigned int>(width / cellWidth);
	cellCountY = static_cast<unsigned int>(height / cellHeight);
	cellCountZ = static_cast<unsigned int>(depth / cellDepth);

	cellAmount = width / cellWidth * height / cellHeight * depth / cellDepth;
	HANDLE_ERROR(hipMalloc((void**)&gridCellIds, objectsCount * sizeof(unsigned int)));
	HANDLE_ERROR(hipMalloc((void**)&particleIds, objectsCount * sizeof(unsigned int)));

	//HANDLE_ERROR(hipMalloc((void**)&gridCellStarts, cellAmount * sizeof(unsigned int)));
	//HANDLE_ERROR(hipMalloc((void**)&gridCellEnds, cellAmount * sizeof(unsigned int)));

	treeNodesCount = (pow(8, levels) - 1) / 7;
	printf("Octree nodes count: %d\n", treeNodesCount);
	HANDLE_ERROR(hipMalloc((void**)&masks, treeNodesCount * sizeof(unsigned char)));
	
	lastNonLeafLayerCount = pow(8, levels - 2);
	HANDLE_ERROR(hipMalloc((void**)&treeData, lastNonLeafLayerCount * sizeof(unsigned int)));
}

void OctreeGrid::calculateGrid(const float* positionX, const float* positionY, const float* positionZ, unsigned int particleCount)
{
	createOctreeGridData(positionX, positionY, positionZ, gridCellIds, particleIds, cellWidth,
		cellHeight, cellDepth, /*gridCellStarts, gridCellEnds, */particleCount);


}