#include "hip/hip_runtime.h"
#include "vein_collisions.cuh"
#include "../utilities/vertex_index_enum.h"

namespace sim
{
	__device__ ray::ray(float3 origin, float3 direction) : origin(origin), direction(direction) {}

	// 1. Calculate collisions between particles and vein triangles
	// 2. Propagate forces into velocities and velocities into positions. Reset forces to 0 afterwards
	__global__ void detectVeinCollisionsAndPropagateParticles(BloodCells cells, DeviceTriangles triangles)
	{
		int part_index = blockDim.x * blockIdx.x + threadIdx.x;

		if (part_index >= cells.particleCount)
			return;

		// propagate force into velocities
		float3 F = cells.particles.force.get(part_index);
		float3 velocity = cells.particles.velocity.get(part_index);
		float3 pos = cells.particles.position.get(part_index);

		// upper and lower bound
		if (pos.y >= 0.9f * height)
			velocity.y -= 5;

		if (pos.y <= 0.1f * height)
			velocity.y += 5;

		velocity = velocity + dt * F;
		float3 velocityDir = normalize(velocity);
		ray r(pos, velocityDir);
		float3 reflectedVelociy;

		// collisions with vein cylinder
		// TODO: this is a naive (no grid) implementation
		if (
			calculateSideCollisions(pos, r, reflectedVelociy, triangles) &&
			length(pos - (pos + r.t * r.direction)) <= 5.0f)
		{
			// triangles move vector, 2 is experimentall constant
			float3 ds = 2 * velocityDir;

			float speed = length(velocity);
			velocity = velocityCollisionDamping * speed * reflectedVelociy;
			// move triangle a bit
			triangles.add(r.objectIndex, vertex0, ds);
			triangles.add(r.objectIndex, vertex1, ds);
			triangles.add(r.objectIndex, vertex2, ds);
		}

		cells.particles.velocity.set(part_index, velocity);

		// propagate velocities into positions
		cells.particles.position.add(part_index, dt * velocity);

		// zero forces
		cells.particles.force.set(part_index, make_float3(0, 0, 0));
	}

	// Calculate whether a collision between a particle (represented by the ray) and a vein triangle occurred
	__device__ bool calculateSideCollisions(float3 position, ray& velocityRay, float3& reflectionVector, DeviceTriangles& triangles)
	{
		constexpr float EPS = 0.000001f;
		for (int i = 0; i < triangles.triangleCount; ++i)
		{
			// triangle vectices and edges
			float3 v1 = triangles.get(i, vertex0);
			float3 v2 = triangles.get(i, vertex1);
			float3 v3 = triangles.get(i, vertex2);
			const float3 edge1 = v2 - v1;
			const float3 edge2 = v3 - v1;

			const float3 h = cross(velocityRay.direction, edge2);
			const float a = dot(edge1, h);
			if (a > -EPS && a < EPS)
				continue; // ray parallel to triangle
			
			const float f = 1 / a;
			const float3 s = velocityRay.origin - v1;
			const float u = f * dot(s, h);
			if (u < 0 || u > 1)
				continue;
			const float3 q = cross(s, edge1);
			const float v = f * dot(velocityRay.direction, q);
			if (v < 0 || u + v > 1)
				continue;
			const float t = f * dot(edge2, q);
			if (t > EPS)
			{
				velocityRay.t = t;
				velocityRay.objectIndex = i;

				// this normal is oriented to the vein interior
				// it is caused by the order of vertices in triangles used to correct face culling
				// change order of edge2 and edge1 in cross product for oposite normal
				// Question: Is the situation when we should use oposite normal possible ?
				float3 normal = normalize(cross(edge2, edge1));
				reflectionVector = velocityRay.direction - 2 * dot(velocityRay.direction, normal) * normal;
				return true;
			}
		}
		return false;
	}
}