#include "hip/hip_runtime.h"
#include "vein_collisions.cuh"
#include "../utilities/vertex_index_enum.h"


namespace sim
{
	__device__ ray::ray(float3 origin, float3 direction) : origin(origin), direction(direction) {}

	__device__ bool realCollisionDetection(float3 v0, float3 v1, float3 v2, ray& velocityRay, float3& reflectionVector)
	{
		constexpr float EPS = 0.000001f;
		const float3 edge1 = v1 - v0;
		const float3 edge2 = v2 - v0;

		const float3 h = cross(velocityRay.direction, edge2);
		const float a = dot(edge1, h);
		if (a > -EPS && a < EPS)
			return false; // ray parallel to triangle

		const float f = 1 / a;
		const float3 s = velocityRay.origin - v0;
		const float u = f * dot(s, h);
		if (u < 0 || u > 1)
			return false;
		const float3 q = cross(s, edge1);
		const float v = f * dot(velocityRay.direction, q);
		if (v < 0 || u + v > 1)
			return false;
		const float t = f * dot(edge2, q);
		if (t > EPS)
		{
			velocityRay.t = t;

			// this normal is oriented to the vein interior
			// it is caused by the order of vertices in triangles used to correct face culling
			// change order of edge2 and edge1 in cross product for oposite normal
			// Question: Is the situation when we should use oposite normal possible ?
			float3 normal = normalize(cross(edge2, edge1));
			reflectionVector = velocityRay.direction - 2 * dot(velocityRay.direction, normal) * normal;
			return true;
		}
	}

	__device__ float3 calculateBaricentric(float3 point, float3 v0, float3 v1, float3 v2)
	{
		float3 baricentric;
		float3 e0 = v1 - v0, e1 = v2 - v1, e2 = point - v0;
		float d00 = dot(e0, e0);
		float d01 = dot(e0, e1);
		float d11 = dot(e1, e1);
		float d20 = dot(e2, e0);
		float d21 = dot(e2, e1);
		float denom = d00 * d11 - d01 * d01;
		baricentric.x = (d11 * d20 - d01 * d21) / denom;
		baricentric.y = (d00 * d21 - d01 * d20) / denom;
		baricentric.z = 1.0f - baricentric.x - baricentric.y;
		return baricentric;
	}



	// 1. Calculate collisions between particles and vein triangles
	// 2. Propagate forces into velocities and velocities into positions. Reset forces to 0 afterwards
	template<>
	__global__ void detectVeinCollisionsAndPropagateParticles<UniformGrid>(BloodCells cells, DeviceTriangles triangles, UniformGrid triangleGrid )
	{
		int particleId = blockDim.x * blockIdx.x + threadIdx.x;

		if (particleId >= cells.particleCount)
			return;

		// propagate force into velocities
		float3 F = cells.particles.force.get(particleId);
		float3 velocity = cells.particles.velocity.get(particleId);
		float3 pos = cells.particles.position.get(particleId);

		// upper and lower bound
		if (pos.y >= 0.9f * height)
			velocity.y -= 5;

		if (pos.y <= 0.1f * height)
			velocity.y += 5;

		velocity = velocity + dt * F;
		float3 velocityDir = normalize(velocity);
		ray r(pos, velocityDir);
		float3 reflectedVelociy = make_float3(0, 0, 0);

		bool collicionOccured = false;
		int xId = static_cast<unsigned int>(cells.particles.position.x[particleId] / cellWidth);
		int yId = static_cast<unsigned int>(cells.particles.position.y[particleId] / cellHeight) * cellCountX;
		int zId = static_cast<unsigned int>(cells.particles.position.z[particleId] / cellDepth) * cellCountX * cellCountY;

		// collisions with vein cylinder
		if (xId < 1)
		{
			if (yId < 1)
			{
				if (zId < 1)
				{
					collicionOccured = calculateSideCollisions<0, 1, 0, 1, 0, 1>(pos, r, reflectedVelociy, triangles, triangleGrid );
				}
				else if (zId > cellCountZ - 2)
				{
					collicionOccured = calculateSideCollisions<0, 1, 0, 1, -1, 0>(pos, r, reflectedVelociy, triangles, triangleGrid );
				}
				else
				{
					collicionOccured = calculateSideCollisions<0, 1, 0, 1, -1, 1>(pos, r, reflectedVelociy, triangles, triangleGrid );
				}
			}
			else if (yId > cellCountY - 2)
			{
				if (zId < 1)
				{
					collicionOccured = calculateSideCollisions<0, 1, -1, 0, 0, 1>(pos, r, reflectedVelociy, triangles, triangleGrid );
				}
				else if (zId > cellCountZ - 2)
				{
					collicionOccured = calculateSideCollisions<0, 1, -1, 0, -1, 0>(pos, r, reflectedVelociy, triangles, triangleGrid );
				}
				else
				{
					collicionOccured = calculateSideCollisions<0, 1, -1, 0, -1, 1>(pos, r, reflectedVelociy, triangles, triangleGrid );
				}
			}
			else
			{
				if (zId < 1)
				{
					collicionOccured = calculateSideCollisions<0, 1, -1, 1, 0, 1>(pos, r, reflectedVelociy, triangles, triangleGrid );
				}
				else if (zId > cellCountZ - 2)
				{
					collicionOccured = calculateSideCollisions<0, 1, -1, 1, -1, 0>(pos, r, reflectedVelociy, triangles, triangleGrid );
				}
				else
				{
					collicionOccured = calculateSideCollisions<0, 1, -1, 1, -1, 1>(pos, r, reflectedVelociy, triangles, triangleGrid );
				}
			}
		}
		else if (xId > cellCountX - 2)
		{
			if (yId < 1)
			{
				if (zId < 1)
				{
					collicionOccured = calculateSideCollisions<-1, 0, 0, 1, 0, 1>(pos, r, reflectedVelociy, triangles, triangleGrid );
				}
				else if (zId > cellCountZ - 2)
				{
					collicionOccured = calculateSideCollisions<-1, 0, 0, 1, -1, 0>(pos, r, reflectedVelociy, triangles, triangleGrid );
				}
				else
				{
					collicionOccured = calculateSideCollisions<-1, 0, 0, 1, -1, 1>(pos, r, reflectedVelociy, triangles, triangleGrid );
				}
			}
			else if (yId > cellCountY - 2)
			{
				if (zId < 1)
				{
					collicionOccured = calculateSideCollisions<-1, 0, -1, 0, 0, 1>(pos, r, reflectedVelociy, triangles, triangleGrid );
				}
				else if (zId > cellCountZ - 2)
				{
					collicionOccured = calculateSideCollisions<-1, 0, -1, 0, -1, 0>(pos, r, reflectedVelociy, triangles, triangleGrid );
				}
				else
				{
					collicionOccured = calculateSideCollisions<-1, 0, -1, 0, -1, 1>(pos, r, reflectedVelociy, triangles, triangleGrid );
				}
			}
			else
			{
				if (zId < 1)
				{
					collicionOccured = calculateSideCollisions<-1, 0, -1, 1, 0, 1>(pos, r, reflectedVelociy, triangles, triangleGrid );
				}
				else if (zId > cellCountZ - 2)
				{
					collicionOccured = calculateSideCollisions<-1, 0, -1, 1, -1, 0>(pos, r, reflectedVelociy, triangles, triangleGrid );
				}
				else
				{
					collicionOccured = calculateSideCollisions<-1, 0, -1, 1, -1, 1>(pos, r, reflectedVelociy, triangles, triangleGrid );
				}
			}
		}
		else
		{
			if (yId < 1)
			{
				if (zId < 1)
				{
					collicionOccured = calculateSideCollisions<-1, 1, 0, 1, 0, 1>(pos, r, reflectedVelociy, triangles, triangleGrid );
				}
				else if (zId > cellCountZ - 2)
				{
					collicionOccured = calculateSideCollisions<-1, 1, 0, 1, -1, 0>(pos, r, reflectedVelociy, triangles, triangleGrid );
				}
				else
				{
					collicionOccured = calculateSideCollisions<-1, 1, 0, 1, -1, 1>(pos, r, reflectedVelociy, triangles, triangleGrid );
				}
			}
			else if (yId > cellCountY - 2)
			{
				if (zId < 1)
				{
					collicionOccured = calculateSideCollisions<-1, 1, -1, 0, 0, 1>(pos, r, reflectedVelociy, triangles, triangleGrid );
				}
				else if (zId > cellCountZ - 2)
				{
					collicionOccured = calculateSideCollisions<-1, 1, -1, 0, -1, 0>(pos, r, reflectedVelociy, triangles, triangleGrid );
				}
				else
				{
					collicionOccured = calculateSideCollisions<-1, 1, -1, 0, -1, 1>(pos, r, reflectedVelociy, triangles, triangleGrid );
				}
			}
			else
			{
				if (zId < 1)
				{
					collicionOccured = calculateSideCollisions<-1, 1, -1, 1, 0, 1>(pos, r, reflectedVelociy, triangles, triangleGrid );
				}
				else if (zId > cellCountZ - 2)
				{
					collicionOccured = calculateSideCollisions<-1, 1, -1, 1, -1, 0>(pos, r, reflectedVelociy, triangles, triangleGrid );
				}
				else
				{
					collicionOccured = calculateSideCollisions<-1, 1, -1, 1, -1, 1>(pos, r, reflectedVelociy, triangles, triangleGrid );
				}
			}
		}

		if (collicionOccured)
		{
			// triangles move vector, 2 is experimentall constant
			float3 ds = 0.8f * velocityDir;

			float speed = length(velocity);
			velocity = velocityCollisionDamping * speed * reflectedVelociy;

			float3 v0 = triangles.get(r.objectIndex, vertex0);
			float3 v1 = triangles.get(r.objectIndex, vertex1);
			float3 v2 = triangles.get(r.objectIndex, vertex2);
			float3 baricentric = calculateBaricentric(pos + r.t * r.direction, v0, v1, v2);

			// move triangle a bit
			triangles.add(r.objectIndex, vertex0, baricentric.x * ds);
			triangles.add(r.objectIndex, vertex1, baricentric.y * ds);
			triangles.add(r.objectIndex, vertex2, baricentric.z * ds);
		}

		cells.particles.velocity.set(particleId, velocity);

		// propagate velocities into positions
		cells.particles.position.add(particleId, dt * velocity);

		// zero forces
		cells.particles.force.set(particleId, make_float3(0, 0, 0));
	}

	// 1. Calculate collisions between particles and vein triangles
	// 2. Propagate forces into velocities and velocities into positions. Reset forces to 0 afterwards
	template<>
	__global__ void detectVeinCollisionsAndPropagateParticles<NoGrid>(BloodCells cells, DeviceTriangles triangles, NoGrid  triangleGrid )
	{
		int particleId = blockDim.x * blockIdx.x + threadIdx.x;

		if (particleId >= cells.particleCount)
			return;

		// propagate force into velocities
		float3 F = cells.particles.force.get(particleId);
		float3 velocity = cells.particles.velocity.get(particleId);
		float3 pos = cells.particles.position.get(particleId);

		// upper and lower bound
		if (pos.y >= 0.9f * height)
			velocity.y -= 5;

		if (pos.y <= 0.1f * height)
			velocity.y += 5;

		velocity = velocity + dt * F;
		float3 velocityDir = normalize(velocity);
		ray r(pos, velocityDir);
		float3 reflectedVelociy = make_float3(0, 0, 0);

		bool collicionOccured = false;

		for (int triangleId = 0; triangleId < triangles.triangleCount; ++triangleId)
		{
			// triangle vectices and edges
			float3 v0 = triangles.get(triangleId, vertex0);
			float3 v1 = triangles.get(triangleId, vertex1);
			float3 v2 = triangles.get(triangleId, vertex2);

			if (!(realCollisionDetection(v0, v1, v2, r, reflectedVelociy) 
				&& length(pos - (pos + r.t * r.direction)) <= veinImpactDistance))
				continue;

			r.objectIndex = triangleId;
			collicionOccured = true;
			break;
		}

		if (collicionOccured)
		{
			// triangles move vector, 2 is experimentall constant
			float3 ds = 0.8f * velocityDir;

			float speed = length(velocity);
			velocity = velocityCollisionDamping * speed * reflectedVelociy;

			float3 v0 = triangles.get(r.objectIndex, vertex0);
			float3 v1 = triangles.get(r.objectIndex, vertex1);
			float3 v2 = triangles.get(r.objectIndex, vertex2);
			float3 baricentric = calculateBaricentric(pos + r.t * r.direction, v0, v1, v2);

			// move triangle a bit
			triangles.add(r.objectIndex, vertex0, baricentric.x * ds);
			triangles.add(r.objectIndex, vertex1, baricentric.y * ds);
			triangles.add(r.objectIndex, vertex2, baricentric.z * ds);
		}

		cells.particles.velocity.set(particleId, velocity);

		// propagate velocities into positions
		cells.particles.position.add(particleId, dt * velocity);

		// zero forces
		cells.particles.force.set(particleId, make_float3(0, 0, 0));
	}

	template<int xMin, int xMax, int yMin, int yMax, int zMin, int zMax>
	__device__ bool calculateSideCollisions(float3 position, ray& velocityRay, float3& reflectionVector, DeviceTriangles& triangles, UniformGrid& triangleGrid )
	{
		unsigned int cellId = triangleGrid.calculateCellId(position);

		#pragma unroll
		for (int x = xMin; x <= xMax; x++)
		{
			#pragma unroll	
			for (int y = yMin; y <= yMax; y++)
			{
				#pragma unroll
				for (int z = zMin; z <= zMax; z++)
				{
					int neighborCellId = cellId + z * cellCountX * cellCountY + y * cellCountX + x;

					for (int i = triangleGrid.gridCellStarts[neighborCellId]; i <= triangleGrid.gridCellEnds[neighborCellId]; i++)
					{
						// triangle vectices and edges
						unsigned int triangleId = triangleGrid.particleIds[i];
						float3 v0 = triangles.get(triangleId, vertex0);
						float3 v1 = triangles.get(triangleId, vertex1);
						float3 v2 = triangles.get(triangleId, vertex2);

						if (!(realCollisionDetection(v0, v1, v2, velocityRay, reflectionVector) 
							&& length(position - (position + velocityRay.t * velocityRay.direction)) <= veinImpactDistance))
							continue;

						velocityRay.objectIndex = triangleId;
						return true;
					}
				}
			}
		}
		return false;
	}
}