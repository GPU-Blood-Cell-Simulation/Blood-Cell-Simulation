#include "hip/hip_runtime.h"
#include "vein_collisions.cuh"
#include "../utilities/vertex_index_enum.h"

namespace sim
{
	__device__ ray::ray(float3 origin, float3 direction) : origin(origin), direction(direction) {}

	// 1. Calculate collisions between particles and vein triangles
	// 2. Propagate forces into velocities and velocities into positions. Reset forces to 0 afterwards
	__global__ void detectVeinCollisionsAndPropagateParticles(BloodCells cells, DeviceTriangles triangles)
	{
		int partIndex = blockDim.x * blockIdx.x + threadIdx.x;

		if (partIndex >= cells.particleCount)
			return;

		float3 F = cells.particles.force.get(partIndex);
		float3 velocity = cells.particles.velocity.get(partIndex);
		float3 pos = cells.particles.position.get(partIndex);

		// upper and lower bound
		if (pos.y >= 0.9f * height)
			velocity.y -= 5;

		if (pos.y <= 0.1f * height)
			velocity.y += 5;

		// propagate particle forces into velocities
		velocity = velocity + dt * F;
		float3 velocityDir = normalize(velocity);
		ray r(pos, velocityDir);
		float3 reflectedVelociy;

		// collisions with vein cylinder
		// TODO: this is a naive (no grid) implementation
		if (
			calculateSideCollisions(pos, r, reflectedVelociy, triangles) &&
			length_squared(pos - (pos + r.t * r.direction)) <= 25.0f)
		{
			// triangles move vector, 2 is experimentall constant
			float3 ds = 0.8f * velocityDir;

			float speed = length(velocity);
			velocity = velocityCollisionDamping * speed * reflectedVelociy;

			unsigned int vertexIndex0 = triangles.getIndex(r.objectIndex, vertex0);
			unsigned int vertexIndex1 = triangles.getIndex(r.objectIndex, vertex1);
			unsigned int vertexIndex2 = triangles.getIndex(r.objectIndex, vertex2);

			float3 v0 = triangles.position.get(vertexIndex0);
			float3 v1 = triangles.position.get(vertexIndex1);
			float3 v2 = triangles.position.get(vertexIndex2);
			float3 baricentric = calculateBaricentric(pos + r.t * r.direction, v0, v1, v2);

			// move triangle a bit
			triangles.force.add(vertexIndex0, baricentric.x * ds);
			triangles.force.add(vertexIndex1, baricentric.y * ds);
			triangles.force.add(vertexIndex2, baricentric.z * ds);

		}

		cells.particles.velocity.set(partIndex, velocity);

		// propagate velocities into positions
		cells.particles.position.add(partIndex, dt * velocity);

		// zero forces
		cells.particles.force.set(partIndex, make_float3(0, 0, 0));
	}

	// Calculate whether a collision between a particle (represented by the ray) and a vein triangle occurred
	__device__ bool calculateSideCollisions(float3 position, ray& velocityRay, float3& reflectionVector, DeviceTriangles& triangles)
	{
		constexpr float EPS = 0.000001f;
		for (int i = 0; i < triangles.triangleCount; ++i)
		{
			// triangle vectices and edges
			float3 v0 = triangles.position.get(triangles.getIndex(i, vertex0));
			float3 v1 = triangles.position.get(triangles.getIndex(i, vertex1));
			float3 v2 = triangles.position.get(triangles.getIndex(i, vertex2));
			const float3 edge1 = v1 - v0;
			const float3 edge2 = v2 - v0;

			const float3 h = cross(velocityRay.direction, edge2);
			const float a = dot(edge1, h);
			if (a > -EPS && a < EPS)
				continue; // ray parallel to triangle
			
			const float f = 1 / a;
			const float3 s = velocityRay.origin - v0;
			const float u = f * dot(s, h);
			if (u < 0 || u > 1)
				continue;
			const float3 q = cross(s, edge1);
			const float v = f * dot(velocityRay.direction, q);
			if (v < 0 || u + v > 1)
				continue;
			const float t = f * dot(edge2, q);
			if (t > EPS)
			{
				velocityRay.t = t;
				velocityRay.objectIndex = i;

				// this normal is oriented to the vein interior
				// it is caused by the order of vertices in triangles used to correct face culling
				// change order of edge2 and edge1 in cross product for oposite normal
				// Question: Is the situation when we should use oposite normal possible ?
				float3 normal = normalize(cross(edge2, edge1));
				reflectionVector = velocityRay.direction - 2 * dot(velocityRay.direction, normal) * normal;
				return true;
			}
		}
		return false;
	}

	__device__ float3 calculateBaricentric(float3 point, float3 v0, float3 v1, float3 v2)
	{
		float3 baricentric;
		float3 e0 = v1 - v0, e1 = v2 - v1, e2 = point - v0;
		float d00 = dot(e0, e0);
		float d01 = dot(e0, e1);
		float d11 = dot(e1, e1);
		float d20 = dot(e2, e0);
		float d21 = dot(e2, e1);
		float denom = d00 * d11 - d01 * d01;
		baricentric.x = (d11 * d20 - d01 * d21) / denom;
		baricentric.y = (d00 * d21 - d01 * d20) / denom;
		baricentric.z = 1.0f - baricentric.x - baricentric.y;
		return baricentric;
	}

	/// <summary>
	/// Propagates the forces at vein triangle indices into their neighbors using elastic springs
	/// </summary>
	/// <param name="triangles"></param>
	/// <returns></returns>
	__global__ void propagateVeinTriangleVertices(DeviceTriangles triangles)
	{
		int vertex = blockDim.x * blockIdx.x + threadIdx.x;

		if (vertex >= triangles.vertexCount)
			return;

		// propagate forces into velocities
		triangles.velocity.add(vertex, dt * triangles.force.get(vertex));

		// propagate velocities into positions
		triangles.position.add(vertex, dt * triangles.velocity.get(vertex));

		// zero forces
		triangles.force.set(vertex, make_float3(0, 0, 0));
	}
}