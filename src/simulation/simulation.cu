#include "hip/hip_runtime.h"
#include "simulation.cuh"
#include "../defines.hpp"
#include "vein_collisions.cuh"
#include "particle_collisions.cuh"
#include "../utilities/cuda_handle_error.cuh"

#include <cmath>
#include <ctime>

#include "hip/hip_runtime.h"
#include ""


namespace sim
{
	__global__ void setupCurandStatesKernel(hiprandState* states, const unsigned long seed, const int particleCount);

	__global__ void generateRandomPositionsKernel(hiprandState* states, Particles particles, const int particleCount);

	// Generate initial positions and velocities of particles
	void generateRandomPositions(Particles& particles, const int particleCount)
	{
		int threadsPerBlock = particleCount > 1024 ? 1024 : particleCount;
		int blocks = (particleCount + threadsPerBlock - 1) / threadsPerBlock;

		// Set up random seeds
		hiprandState* devStates;
		HANDLE_ERROR(hipMalloc(&devStates, particleCount * sizeof(hiprandState)));
		srand(static_cast<unsigned int>(time(0)));
		int seed = rand();
		setupCurandStatesKernel << <blocks, threadsPerBlock >> > (devStates, seed, particleCount);

		// Generate random positions and velocity vectors

		generateRandomPositionsKernel << <blocks, threadsPerBlock >> > (devStates, particles, particleCount);

		HANDLE_ERROR(hipFree(devStates));
	}

	__global__ void setupCurandStatesKernel(hiprandState* states, const unsigned long seed, const int particleCount)
	{
		int id = blockIdx.x * blockDim.x + threadIdx.x;
		if (id >= particleCount)
			return;
		hiprand_init(seed, id, 0, &states[id]);
	}

	// Generate random positions and velocities at the beginning
	__global__ void generateRandomPositionsKernel(hiprandState* states, Particles p, const int particleCount)
	{
		int id = blockIdx.x * blockDim.x + threadIdx.x;
		if (id >= particleCount)
			return;

		p.position.x[id] = hiprand_uniform(&states[id]) * width / 3 + width/6;
		p.position.y[id] = hiprand_uniform(&states[id]) * height / 3 + height/6;
		p.position.z[id] = hiprand_uniform(&states[id]) * depth / 3 + depth/6;

		p.force.x[id] = 0;
		p.force.y[id] = 0;
		p.force.z[id] = 0;
	}

	// Main simulation function, called every frame
	void calculateNextFrame(BloodCells& bloodCells, DeviceTriangles& triangles, Grid particleGrid, Grid triangleGrid, unsigned int triangleCount)
	{
		std::visit([&](auto&& g1, auto&& g2)
			{
				// 1. Calculate grids
				g1->calculateGrid(bloodCells.particles, bloodCells.particleCount);
				g2->calculateGrid(triangles.centers.x, triangles.centers.y, triangles.centers.z, triangleCount);

				int threadsPerBlock = bloodCells.particleCount > 1024 ? 1024 : bloodCells.particleCount;
				int blDim = std::ceil(float(bloodCells.particleCount) / threadsPerBlock);

				// 2. Detect particle collisions
				calculateParticleCollisions << < dim3(blDim), threadsPerBlock >> > (bloodCells, *g1);
				//HANDLE_ERROR(hipPeekAtLastError());
				// 3. Propagate forces into neighbors
				bloodCells.propagateForces();
				//HANDLE_ERROR(hipPeekAtLastError());
				// 4. Detect vein collisions and propagate forces -> velocities, velocities -> positions
				detectVeinCollisionsAndPropagateParticles << < dim3(blDim), threadsPerBlock >> > (bloodCells, triangles, *g1, *g2, FRAME);
				//HANDLE_ERROR(hipPeekAtLastError());
			}, particleGrid, triangleGrid);
	}
}