#include "hip/hip_runtime.h"
#include "simulation.cuh"
#include "../defines.hpp"
#include "vein_collisions.cuh"
#include "particle_collisions.cuh"
#include "../utilities/cuda_handle_error.cuh"

#include <cmath>
#include <ctime>

#include "hip/hip_runtime.h"
#include ""


namespace sim
{
	__global__ void setupCurandStatesKernel(hiprandState* states, const unsigned long seed, const int particleCount);

	__global__ void generateRandomPositionsKernel(hiprandState* states, Particles particles, const int particleCount);

	// Generate initial positions and velocities of particles
	void generateRandomPositions(Particles& particles, const int particleCount)
	{
		int threadsPerBlock = particleCount > 1024 ? 1024 : particleCount;
		int blocks = (particleCount + threadsPerBlock - 1) / threadsPerBlock;

		// Set up random seeds
		hiprandState* devStates;
		HANDLE_ERROR(hipMalloc(&devStates, particleCount * sizeof(hiprandState)));
		srand(static_cast<unsigned int>(time(0)));
		int seed = rand();
		setupCurandStatesKernel << <blocks, threadsPerBlock >> > (devStates, seed, particleCount);

		// Generate random positions and velocity vectors

		generateRandomPositionsKernel << <blocks, threadsPerBlock >> > (devStates, particles, particleCount);

		HANDLE_ERROR(hipFree(devStates));
	}

	__global__ void setupCurandStatesKernel(hiprandState* states, const unsigned long seed, const int particleCount)
	{
		int id = blockIdx.x * blockDim.x + threadIdx.x;
		if (id >= particleCount)
			return;
		hiprand_init(seed, id, 0, &states[id]);
	}

	// Generate random positions and velocities at the beginning
	__global__ void generateRandomPositionsKernel(hiprandState* states, Particles p, const int particleCount)
	{
		int id = blockIdx.x * blockDim.x + threadIdx.x;
		if (id >= particleCount)
			return;

		p.position.x[id] = hiprand_uniform(&states[id]) * width / 3 + width/6;
		p.position.y[id] = hiprand_uniform(&states[id]) * height / 3 + height/6;
		p.position.z[id] = hiprand_uniform(&states[id]) * depth / 3 + depth/6;

		p.force.x[id] = 0;
		p.force.y[id] = 0;
		p.force.z[id] = 0;
	}

	// Main simulation function, called every frame
	void calculateNextFrame(BloodCells& bloodCells, DeviceTriangles& triangles, Grid grid, UniformGrid triangleGrid, unsigned int triangleCount)
	{
		// 1. Calculate grid
		std::visit([&](auto&& g)
			{
				g->calculateGrid(bloodCells.particles, bloodCells.particleCount);
			}, grid);
		triangleGrid.calculateGrid(triangles.centers.x, triangles.centers.y, triangles.centers.z, triangleCount);

		int threadsPerBlock = bloodCells.particleCount > 1024 ? 1024 : bloodCells.particleCount;
		int blDim = std::ceil(float(bloodCells.particleCount) / threadsPerBlock);
		
		// 2. Detect particle collisions
		std::visit([&](auto&& g)
			{
				calculateParticleCollisions << < dim3(blDim), threadsPerBlock >> > (bloodCells, *g);
			}, grid);
		

		// 3. Propagate forces into neighbors

		bloodCells.propagateForces();

		// 4. Detect vein collisions and propagate forces -> velocities, velocities -> positions

		detectVeinCollisionsAndPropagateParticles << < dim3(blDim), threadsPerBlock >> > (bloodCells, triangles, triangleGrid, triangleGrid.cellAmount);
	}
}