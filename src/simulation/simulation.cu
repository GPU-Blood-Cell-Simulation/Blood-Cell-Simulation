#include "hip/hip_runtime.h"
#include "simulation.cuh"
#include "../defines.hpp"
#include "vein_collisions.cuh"
#include "particle_collisions.cuh"
#include "../utilities/cuda_handle_error.cuh"

#include <cmath>
#include <ctime>

#include "hip/hip_runtime.h"
#include ""


namespace sim
{
	__global__ void setupCurandStatesKernel(hiprandState* states, const unsigned long seed, const int particleCount);

	__global__ void generateRandomPositionsKernel(hiprandState* states, Particles particles, const int particleCount, glm::vec3 cylinderBaseCenter/*, float cylinderRadius, float cylinderHeight*/);

	SimulationController::SimulationController(BloodCells& bloodCells, DeviceTriangles& triangles, Grid grid) : bloodCells(bloodCells), triangles(triangles), grid(grid)
	{
		// Generate random particle positions
		generateRandomPositions();
	}

	// Generate initial positions and velocities of particles
	void SimulationController::generateRandomPositions()
	{
		int threadsPerBlock = bloodCells.particleCount > 1024 ? 1024 : bloodCells.particleCount;
		int blocks = (bloodCells.particleCount + threadsPerBlock - 1) / threadsPerBlock;

		// Set up random seeds
		hiprandState* devStates;
		HANDLE_ERROR(hipMalloc(&devStates, bloodCells.particleCount * sizeof(hiprandState)));
		srand(static_cast<unsigned int>(time(0)));
		int seed = rand();
		setupCurandStatesKernel << <blocks, threadsPerBlock >> > (devStates, seed, bloodCells.particleCount);

		// Generate random positions and velocity vectors

		generateRandomPositionsKernel << <blocks, threadsPerBlock >> > (devStates, bloodCells.particles, bloodCells.particleCount, cylinderBaseCenter);


		HANDLE_ERROR(hipFree(devStates));
	}

	__global__ void setupCurandStatesKernel(hiprandState* states, const unsigned long seed, const int particleCount)
	{
		int id = blockIdx.x * blockDim.x + threadIdx.x;
		if (id >= particleCount)
			return;
		hiprand_init(seed, id, 0, &states[id]);
	}

	// Generate random positions and velocities at the beginning
	__global__ void generateRandomPositionsKernel(hiprandState* states, Particles p, const int particleCount, glm::vec3 cylinderBaseCenter/*, float cylinderRadius, float cylinderHeight*/)
	{
		int id = blockIdx.x * blockDim.x + threadIdx.x;
		if (id >= particleCount)
			return;

		p.position.x[id] = cylinderBaseCenter.x - cylinderRadius * 0.5f + hiprand_uniform(&states[id]) * cylinderRadius;
		p.position.y[id] = cylinderBaseCenter.y - cylinderRadius * 0.5f + hiprand_uniform(&states[id]) * cylinderRadius + cylinderHeight/2;
		p.position.z[id] = cylinderBaseCenter.z - cylinderRadius * 0.5f + hiprand_uniform(&states[id]) * cylinderRadius;

		p.force.x[id] = 0;
		p.force.y[id] = 0;
		p.force.z[id] = 0;
	}

	// Main simulation function, called every frame
	void SimulationController::calculateNextFrame()
	{
    // 1. Calculate grids
		std::visit([&](auto&& g1, auto&& g2)
			{
				// 1. Calculate grids
				g1->calculateGrid(bloodCells.particles, bloodCells.particleCount);
				g2->calculateGrid(triangles.centers.x, triangles.centers.y, triangles.centers.z, triangleCount);

				// anything above 768 threads (25 warps) trigger an error
				// 'too many resources requested for launch'
				// maybe possible to solve
				int threadsPerBlock = bloodCells.particleCount > 768 ? 768 : bloodCells.particleCount;
				int blDim = std::ceil(float(bloodCells.particleCount) / threadsPerBlock);

				// 2. Detect particle collisions
				calculateParticleCollisions << < dim3(blDim), threadsPerBlock >> > (bloodCells, *g1);
				HANDLE_ERROR(hipPeekAtLastError());

				// 3. Propagate particle forces into neighbors

        bloodCells.propagateForces();
        HANDLE_ERROR(hipPeekAtLastError());
    
        // 4. Detect vein collisions and propagate forces -> velocities, velocities -> positions

        detectVeinCollisionsAndPropagateParticles << < dim3(blDim), threadsPerBlock >> > (bloodCells, triangles);
        HANDLE_ERROR(hipPeekAtLastError());
    
        // 5. Gather forces from neighbors

        triangles.gatherForcesFromNeighbors();
        HANDLE_ERROR(hipPeekAtLastError());
    
        // 6. Propagate forces -> velocities, velocities -> positions for vein triangles
        triangles.propagateForcesIntoPositions();
        HANDLE_ERROR(hipPeekAtLastError());
    
				// 5. Recalculate triangles centers
				triangles.calculateCenters();
				HANDLE_ERROR(hipPeekAtLastError());

			}, particleGrid, triangleGrid);
	}
}