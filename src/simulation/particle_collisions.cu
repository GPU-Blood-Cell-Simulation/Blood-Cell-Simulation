#include "hip/hip_runtime.h"
#include "particle_collisions.cuh"
#include "../utilities/math.cuh"


namespace sim
{
	// Calculate collisions between particles
	__global__ void detectParticleCollisions(BloodCells cells, unsigned int* gridCellIds, unsigned int* particleIds,
		unsigned int* gridCellStarts, unsigned int* gridCellEnds)
	{
		int id = blockIdx.x * blockDim.x + threadIdx.x;
		if (id >= cells.particleCount)
			return;

		int particleId = particleIds[id];
		float3 p1 = cells.particles.position.get(particleId);


		// Naive implementation
		/*for (int i = 0; i < particleCount; i++)
		{
			if (id == i || i == secondParticle)
				continue;

			float3 p2 = particles.position.get(i);
			if (length(p1 - p2) <= 5.0f)
			{
				particles.force.set(id, 50.0f * normalize(p1 - p2));
			}
		}*/

		// Using uniform grid

		int cellId = gridCellIds[id];

		for (int i = gridCellStarts[cellId]; i <= gridCellEnds[cellId]; i++)
		{
			int secondParticleId = particleIds[i];
			if (particleId == secondParticleId)
				continue;

			float3 p2 = cells.particles.position.get(secondParticleId);
			if (length(p1 - p2) <= 5.0f)
			{
				// Uncoalesced writes - area for optimization
				cells.particles.force.set(particleId, 50.0f * normalize(p1 - p2));
			}
		}
	}
}