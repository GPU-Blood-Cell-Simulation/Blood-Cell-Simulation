#include "hip/hip_runtime.h"
#include "simulation_controller.cuh"

#include "../meta_factory/blood_cell_factory.hpp"
#include "../meta_factory/vein_factory.hpp"
#include "../objects/particles.cuh"
#include "particle_collisions.cuh"
#include "../utilities/cuda_handle_error.cuh"
#include "vein_collisions.cuh"

#include <cmath>
#include <ctime>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>


namespace sim
{
	__global__ void setupCurandStatesKernel(hiprandState* states, unsigned long seed);

	__global__ void generateRandomPositionsKernel(hiprandState* states, Particles particles, glm::vec3 cylinderBaseCenter);


	SimulationController::SimulationController(BloodCells& bloodCells, VeinTriangles& triangles, Grid particleGrid, Grid triangleGrid) :
		bloodCells(bloodCells), triangles(triangles), particleGrid(particleGrid), triangleGrid(triangleGrid),
		bloodCellsThreads(particleCount),
		veinVerticesThreads(triangles.vertexCount),
		veinTrianglesThreads(triangles.triangleCount)
	{
		// Create streams
		for (int i = 0; i < bloodCellTypeCount; i++)
		{
			streams[i] = hipStream_t();
			hipStreamCreate(&streams[i]);
		}

		// Generate random particle positions
		generateRandomPositions();
	}

	// Generate initial positions and velocities of particles
	void SimulationController::generateRandomPositions()
	{
		// Set up random seeds
		hiprandState* devStates;
		HANDLE_ERROR(hipMalloc(&devStates, particleCount * sizeof(hiprandState)));
		srand(static_cast<unsigned int>(time(0)));
		int seed = rand();
		setupCurandStatesKernel << <bloodCellsThreads.blocks, bloodCellsThreads.threadsPerBlock >> > (devStates, seed);

		// Generate random positions and velocity vectors

		generateRandomPositionsKernel << <bloodCellsThreads.blocks, bloodCellsThreads.threadsPerBlock >> > (devStates, bloodCells.particles, cylinderBaseCenter);


		HANDLE_ERROR(hipFree(devStates));
	}

	__global__ void setupCurandStatesKernel(hiprandState* states, unsigned long seed)
	{
		int id = blockIdx.x * blockDim.x + threadIdx.x;
		if (id >= particleCount)
			return;
		hiprand_init(seed, id, 0, &states[id]);
	}

	// Generate random positions and velocities at the beginning
	__global__ void generateRandomPositionsKernel(hiprandState* states, Particles particles, glm::vec3 cylinderBaseCenter/*, float cylinderRadius, float cylinderHeight*/)
	{
		int id = blockIdx.x * blockDim.x + threadIdx.x;
		if (id >= particleCount)
			return;

		particles.positions.x[id] = cylinderBaseCenter.x - cylinderRadius * 0.5f + hiprand_uniform(&states[id]) * cylinderRadius;
		particles.positions.y[id] = cylinderBaseCenter.y - cylinderRadius * 0.5f + hiprand_uniform(&states[id]) * cylinderRadius + cylinderHeight/2;
		particles.positions.z[id] = cylinderBaseCenter.z - cylinderRadius * 0.5f + hiprand_uniform(&states[id]) * cylinderRadius;

		particles.velocities.x[id] = 0;
		particles.velocities.y[id] = -10;
		particles.velocities.z[id] = 0;

		particles.forces.x[id] = 0;
		particles.forces.y[id] = 0;
		particles.forces.z[id] = 0;
	}

	// Main simulation function, called every frame
	void SimulationController::calculateNextFrame()
	{
		std::visit([&](auto&& g1, auto&& g2)
			{
				// 1. Calculate grids
				// TODO: possible optimization - these grisds can be calculated simultaneously
				g1->calculateGrid(bloodCells.particles, particleCount);
				g2->calculateGrid(triangles.centers.x, triangles.centers.y, triangles.centers.z, triangles.triangleCount);

				// 2. Detect particle collisions
				calculateParticleCollisions << < bloodCellsThreads.blocks, bloodCellsThreads.threadsPerBlock >> > (bloodCells, *g1);
				HANDLE_ERROR(hipPeekAtLastError());

				// 3. Propagate particle forces into neighbors

				bloodCells.gatherForcesFromNeighbors(streams);
				HANDLE_ERROR(hipPeekAtLastError());
    
				// 4. Detect vein collisions and propagate forces -> velocities, velocities -> positions for particles

				detectVeinCollisionsAndPropagateParticles << < bloodCellsThreads.blocks, bloodCellsThreads.threadsPerBlock >> > (bloodCells, triangles, *g2);
				HANDLE_ERROR(hipPeekAtLastError());
    
				// 5. Propagate triangle forces into neighbors

				triangles.gatherForcesFromNeighbors(veinVerticesThreads.blocks, veinVerticesThreads.threadsPerBlock);
				HANDLE_ERROR(hipPeekAtLastError());
    
				// 6. Propagate forces -> velocities, velocities -> positions for vein triangles
				triangles.propagateForcesIntoPositions(veinVerticesThreads.blocks, veinVerticesThreads.threadsPerBlock);
				HANDLE_ERROR(hipPeekAtLastError());
    
				// 7. Recalculate triangles centers
				triangles.calculateCenters(veinTrianglesThreads.blocks, veinTrianglesThreads.threadsPerBlock);
				HANDLE_ERROR(hipPeekAtLastError());

				/*if constexpr (useBloodFlow)
				{
					endVeinHandler.Handle(bloodCells);
					HANDLE_ERROR(hipPeekAtLastError());
				}		*/

			}, particleGrid, triangleGrid);
	}
}