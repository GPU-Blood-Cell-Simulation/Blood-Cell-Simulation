#include "hip/hip_runtime.h"
#include "simulation_controller.cuh"

#include "../meta_factory/blood_cell_factory.hpp"
#include "../meta_factory/vein_factory.hpp"
#include "../objects/particles.cuh"
#include "particle_collisions.cuh"
#include "../utilities/cuda_handle_error.cuh"
#include "vein_collisions.cuh"
#include "vein_end.cuh"

#include <cmath>
#include <ctime>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <algorithm>

namespace sim
{
	__global__ void setupCurandStatesKernel(hiprandState* states, unsigned long seed);

	template<int bloodCellCount, int particlesInBloodCell, int particlesStart>
	__global__ void generateRandomPositionsKernel(hiprandState* states, Particles particles, glm::vec3 cylinderBaseCenter, cudaVec3 bloodCellModelPosition);


	SimulationController::SimulationController(BloodCells& bloodCells, VeinTriangles& triangles, Grid particleGrid, Grid triangleGrid) :
		bloodCells(bloodCells), triangles(triangles), particleGrid(particleGrid), triangleGrid(triangleGrid),
		bloodCellsThreads(particleCount),
		veinVerticesThreads(triangles.vertexCount),
		veinTrianglesThreads(triangles.triangleCount)
	{
		// Create streams
		for (int i = 0; i < bloodCellTypeCount; i++)
		{
			streams[i] = hipStream_t();
			HANDLE_ERROR(hipStreamCreate(&streams[i]));
		}

		// Generate random particle positions
		generateRandomPositions();
	}

	sim::SimulationController::~SimulationController()
	{
		for (int i = 0; i < bloodCellTypeCount; i++)
		{
			HANDLE_ERROR(hipStreamDestroy(streams[i]));
		}
	}

	// Generate initial positions and velocities of particles
	void SimulationController::generateRandomPositions()
	{
		// Set up random seeds
		hiprandState* devStates;
		HANDLE_ERROR(hipMalloc(&devStates, particleCount * sizeof(hiprandState)));
		srand(static_cast<unsigned int>(time(0)));
		int seed = rand();
		setupCurandStatesKernel << <bloodCellsThreads.blocks, bloodCellsThreads.threadsPerBlock >> > (devStates, seed);

		// Generate random positions and velocity vectors
		cudaVec3* models = new cudaVec3[bloodCellTypeCount];
		using IndexList = mp_iota_c<bloodCellTypeCount>;

		mp_for_each<IndexList>([&](auto i)
		{
			using BloodCellDefinition = mp_at_c<BloodCellList, i>;
			constexpr int particlesStart = particlesStarts[i];

			int modelSize = BloodCellDefinition::ParticlesInCell;
			models[i] = cudaVec3(modelSize);
			
			float* xmodel = new float[modelSize];
			float* ymodel = new float[modelSize];
			float* zmodel = new float[modelSize];

			int i = 0;
			std::for_each(bloodCellModels[i].begin(), bloodCellModels[i].end(), [&](auto& v) {
				xmodel[i] = v.x[i];
				ymodel[i] = v.x[i];
				zmodel[i] = v.x[i];
			});

			HANDLE_ERROR(hipMemcpy(bloodCellModel.x, xmodel, modelSize * sizeof(float), hipMemcpyHostToDevice));
			HANDLE_ERROR(hipMemcpy(bloodCellModel.y, ymodel, modelSize * sizeof(float), hipMemcpyHostToDevice));
			HANDLE_ERROR(hipMemcpy(bloodCellModel.z, zmodel, modelSize * sizeof(float), hipMemcpyHostToDevice));

			delete[] xmodel;
			delete[] ymodel;
			delete[] zmodel;

			CudaThreads threads(BloodCellDefinition::count * BloodCellDefinition::particlesInCell);
			generateRandomPositionsKernel<BloodCellDefinition::count, BloodCellDefinition::particlesInCell, particlesStart>
				<< <threads.blocks, threads.threadsPerBlock, 0, streams[i] >> >(devStates, bloodCells.particles, cylinderBaseCenter, models[i]);
		});
		HANDLE_ERROR(hipDeviceSynchronize());
		HANDLE_ERROR(hipFree(devStates));
	}

	__global__ void setupCurandStatesKernel(hiprandState* states, unsigned long seed)
	{
		int id = blockIdx.x * blockDim.x + threadIdx.x;
		if (id >= particleCount)
			return;
		hiprand_init(seed, id, 0, &states[id]);
	}

	template<int bloodCellCount, int particlesInBloodCell, int particlesStart>
	// Generate random positions and velocities at the beginning
	__global__ void generateRandomPositionsKernel(hiprandState* states, Particles particles, glm::vec3 cylinderBaseCenter, 
		cudaVec3 bloodCellModelPosition/*, unsigned int bloodCellModelSize, float cylinderRadius, float cylinderHeight*/)
	{
		int relativeId = blockIdx.x * blockDim.x + threadIdx.x;
		if (relativeId >= particlesInBloodCell * bloodCellCount)
		return;
		int id = particlesStart + relativeId;

		if (!(id % particlesInBloodCell)) {
			particles.positions.x[id] = cylinderBaseCenter.x - cylinderRadius * 0.5f + hiprand_uniform(&states[id]) * cylinderRadius;
			particles.positions.y[id] = cylinderBaseCenter.y - cylinderRadius * 0.5f + hiprand_uniform(&states[id]) * cylinderRadius + cylinderHeight / 2;
			particles.positions.z[id] = cylinderBaseCenter.z - cylinderRadius * 0.5f + hiprand_uniform(&states[id]) * cylinderRadius;
		}
		else {
			particles.positions.x[id] = particles.positions.x[id / particlesInBloodCell] + bloodCellModelPosition.x[id % particlesInBloodCell] - bloodCellModelPosition.x[0];
			particles.positions.x[id] = particles.positions.x[id / particlesInBloodCell] + bloodCellModelPosition.y[id % particlesInBloodCell] - bloodCellModelPosition.y[0];
			particles.positions.x[id] = particles.positions.x[id / particlesInBloodCell] + bloodCellModelPosition.z[id % particlesInBloodCell] - bloodCellModelPosition.z[0];
		}

		particles.velocities.x[id] = 0;
		particles.velocities.y[id] = -10;
		particles.velocities.z[id] = 0;

		particles.forces.x[id] = 0;
		particles.forces.y[id] = 0;
		particles.forces.z[id] = 0;
	}

	// Main simulation function, called every frame
	void SimulationController::calculateNextFrame()
	{
		std::visit([&](auto&& g1, auto&& g2)
			{
				// 1. Calculate grids
				// TODO: possible optimization - these grisds can be calculated simultaneously
				g1->calculateGrid(bloodCells.particles, particleCount);
				g2->calculateGrid(triangles.centers.x, triangles.centers.y, triangles.centers.z, triangles.triangleCount);

				// 2. Detect particle collisions
				calculateParticleCollisions << < bloodCellsThreads.blocks, bloodCellsThreads.threadsPerBlock >> > (bloodCells, *g1);
				HANDLE_ERROR(hipPeekAtLastError());

				// 3. Propagate particle forces into neighbors

				bloodCells.gatherForcesFromNeighbors(streams);
				HANDLE_ERROR(hipPeekAtLastError());
    
				// 4. Detect vein collisions and propagate forces -> velocities, velocities -> positions for particles

				detectVeinCollisionsAndPropagateParticles << < bloodCellsThreads.blocks, bloodCellsThreads.threadsPerBlock >> > (bloodCells, triangles, *g2);
				HANDLE_ERROR(hipPeekAtLastError());
    
				// 5. Propagate triangle forces into neighbors

				triangles.gatherForcesFromNeighbors(veinVerticesThreads.blocks, veinVerticesThreads.threadsPerBlock);
				HANDLE_ERROR(hipPeekAtLastError());
    
				// 6. Propagate forces -> velocities, velocities -> positions for vein triangles
				triangles.propagateForcesIntoPositions(veinVerticesThreads.blocks, veinVerticesThreads.threadsPerBlock);
				HANDLE_ERROR(hipPeekAtLastError());
    
				// 7. Recalculate triangles centers
				triangles.calculateCenters(veinTrianglesThreads.blocks, veinTrianglesThreads.threadsPerBlock);
				HANDLE_ERROR(hipPeekAtLastError());

				if constexpr (useBloodFlow)
				{
					HandleVeinEnd(bloodCells, streams);
					HANDLE_ERROR(hipPeekAtLastError());
				}

			}, particleGrid, triangleGrid);
	}
}