#include "hip/hip_runtime.h"
#include "simulation_controller.cuh"

#include "../meta_factory/blood_cell_factory.hpp"
#include "../meta_factory/vein_factory.hpp"
#include "../objects/particles.cuh"
#include "particle_collisions.cuh"
#include "../utilities/cuda_handle_error.cuh"
#include "vein_collisions.cuh"
#include "vein_end.cuh"

#include <cmath>
#include <ctime>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <algorithm>

namespace sim
{
	__global__ void setupCurandStatesKernel(hiprandState* states, unsigned long seed);

	template<int bloodCellCount, int particlesInBloodCell, int particlesStart>
	__global__ void generateRandomPositionsKernel(hiprandState* states, Particles particles, glm::vec3 cylinderBaseCenter, cudaVec3 bloodCellModelPosition);


	SimulationController::SimulationController(BloodCells& bloodCells, VeinTriangles& triangles, Grid particleGrid, Grid triangleGrid) :
		bloodCells(bloodCells), triangles(triangles), particleGrid(particleGrid), triangleGrid(triangleGrid),
		bloodCellsThreads(particleCount),
		veinVerticesThreads(triangles.vertexCount),
		veinTrianglesThreads(triangles.triangleCount)
	{
		// Create streams
		for (int i = 0; i < bloodCellTypeCount; i++)
		{
			streams[i] = hipStream_t();
			HANDLE_ERROR(hipStreamCreate(&streams[i]));
		}

		// Generate random particle positions
		generateRandomPositions();
	}

	sim::SimulationController::~SimulationController()
	{
		for (int i = 0; i < bloodCellTypeCount; i++)
		{
			HANDLE_ERROR(hipStreamDestroy(streams[i]));
		}
	}

	// Generate initial positions and velocities of particles
	void SimulationController::generateRandomPositions()
	{
		// Set up random seeds
		hiprandState* devStates;
		HANDLE_ERROR(hipMalloc(&devStates, particleCount * sizeof(hiprandState)));
		srand(static_cast<unsigned int>(time(0)));
		int seed = rand();
		setupCurandStatesKernel << <bloodCellsThreads.blocks, bloodCellsThreads.threadsPerBlock >> > (devStates, seed);
		HANDLE_ERROR(hipDeviceSynchronize());
		// Generate random positions and velocity vectors
		std::vector<cudaVec3> models;
		using IndexList = mp_iota_c<bloodCellTypeCount>;
		mp_for_each<IndexList>([&](auto i)
			{

				using BloodCellDefinition = mp_at_c<BloodCellList, i>;
				constexpr int modelSize = BloodCellDefinition::particlesInCell;
				cudaVec3 g_model = cudaVec3(modelSize);
				std::vector<float> xmodel;
				std::vector<float> ymodel;
				std::vector<float> zmodel;
				using verticeIndexList = mp_iota_c<modelSize>;
				using VerticeList = typename BloodCellDefinition::Vertices;

				mp_for_each<verticeIndexList>([&](auto j)
					{
						xmodel.push_back(mp_at_c<VerticeList, j>::x);
						ymodel.push_back(mp_at_c<VerticeList, j>::y);
						zmodel.push_back(mp_at_c<VerticeList, j>::z);
					});
				HANDLE_ERROR(hipDeviceSynchronize());
				HANDLE_ERROR(hipMemcpy(g_model.x, xmodel.data(), modelSize * sizeof(float), hipMemcpyHostToDevice));
				HANDLE_ERROR(hipMemcpy(g_model.y, ymodel.data(), modelSize * sizeof(float), hipMemcpyHostToDevice));
				HANDLE_ERROR(hipMemcpy(g_model.z, zmodel.data(), modelSize * sizeof(float), hipMemcpyHostToDevice));
				models.push_back(g_model);
			});
		mp_for_each<IndexList>([&](auto i)
		{
			using BloodCellDefinition = mp_at_c<BloodCellList, i>;
			constexpr int particlesStart = particlesStarts[i];
			constexpr int modelSize = BloodCellDefinition::particlesInCell;

			CudaThreads threads(BloodCellDefinition::count * BloodCellDefinition::particlesInCell);
			generateRandomPositionsKernel<BloodCellDefinition::count, BloodCellDefinition::particlesInCell, particlesStart>
				<< <threads.blocks, threads.threadsPerBlock, 0, streams[i] >> >(devStates, bloodCells.particles, cylinderBaseCenter, models[i]);
		});
		HANDLE_ERROR(hipDeviceSynchronize());
		HANDLE_ERROR(hipFree(devStates));
	}

	__global__ void setupCurandStatesKernel(hiprandState* states, unsigned long seed)
	{
		int id = blockIdx.x * blockDim.x + threadIdx.x;
		if (id >= particleCount)
			return;
		hiprand_init(seed, id, 0, &states[id]);
	}

	template<int bloodCellCount, int particlesInBloodCell, int particlesStart>
	// Generate random positions and velocities at the beginning
	__global__ void generateRandomPositionsKernel(hiprandState* states, Particles particles, glm::vec3 cylinderBaseCenter, 
		cudaVec3 bloodCellModelPosition/*, unsigned int bloodCellModelSize, float cylinderRadius, float cylinderHeight*/)
	{
		int relativeId = blockIdx.x * blockDim.x + threadIdx.x;
		if (relativeId >= particlesInBloodCell * bloodCellCount)
		return;
		int id = particlesStart + relativeId;

		if (!(id % particlesInBloodCell)) {
			particles.positions.x[id] = cylinderBaseCenter.x - cylinderRadius * 0.5f + hiprand_uniform(&states[id]) * cylinderRadius;
			particles.positions.y[id] = cylinderBaseCenter.y - cylinderRadius * 0.5f + hiprand_uniform(&states[id]) * cylinderRadius + cylinderHeight / 2;
			particles.positions.z[id] = cylinderBaseCenter.z - cylinderRadius * 0.5f + hiprand_uniform(&states[id]) * cylinderRadius;
		}
		else {
			particles.positions.x[id] = particles.positions.x[(id / particlesInBloodCell)*particlesInBloodCell] + bloodCellModelPosition.x[id % particlesInBloodCell] - bloodCellModelPosition.x[0];
			particles.positions.y[id] = particles.positions.y[(id / particlesInBloodCell)*particlesInBloodCell] + bloodCellModelPosition.y[id % particlesInBloodCell] - bloodCellModelPosition.y[0];
			particles.positions.z[id] = particles.positions.z[(id / particlesInBloodCell)*particlesInBloodCell] + bloodCellModelPosition.z[id % particlesInBloodCell] - bloodCellModelPosition.z[0];
		}

		printf("[%d][%d] particle position: x = %.5f, y = %.5f, z = %.5f\n", id, particlesInBloodCell, particles.positions.x[id], particles.positions.y[id], particles.positions.z[id]);

		particles.velocities.x[id] = 0;
		particles.velocities.y[id] = -10;
		particles.velocities.z[id] = 0;

		particles.forces.x[id] = 0;
		particles.forces.y[id] = 0;
		particles.forces.z[id] = 0;
	}

	// Main simulation function, called every frame
	void SimulationController::calculateNextFrame()
	{
		std::visit([&](auto&& g1, auto&& g2)
			{
				// 1. Calculate grids
				// TODO: possible optimization - these grisds can be calculated simultaneously
				g1->calculateGrid(bloodCells.particles, particleCount);
				g2->calculateGrid(triangles.centers.x, triangles.centers.y, triangles.centers.z, triangles.triangleCount);

				// 2. Detect particle collisions
				calculateParticleCollisions << < bloodCellsThreads.blocks, bloodCellsThreads.threadsPerBlock >> > (bloodCells, *g1);
				HANDLE_ERROR(hipPeekAtLastError());

				// 3. Propagate particle forces into neighbors

				bloodCells.gatherForcesFromNeighbors(streams);
				HANDLE_ERROR(hipPeekAtLastError());
    
				// 4. Detect vein collisions and propagate forces -> velocities, velocities -> positions for particles

				detectVeinCollisionsAndPropagateParticles << < bloodCellsThreads.blocks, bloodCellsThreads.threadsPerBlock >> > (bloodCells, triangles, *g2);
				HANDLE_ERROR(hipPeekAtLastError());
    
				// 5. Propagate triangle forces into neighbors

				triangles.gatherForcesFromNeighbors(veinVerticesThreads.blocks, veinVerticesThreads.threadsPerBlock);
				HANDLE_ERROR(hipPeekAtLastError());
    
				// 6. Propagate forces -> velocities, velocities -> positions for vein triangles
				triangles.propagateForcesIntoPositions(veinVerticesThreads.blocks, veinVerticesThreads.threadsPerBlock);
				HANDLE_ERROR(hipPeekAtLastError());
    
				// 7. Recalculate triangles centers
				triangles.calculateCenters(veinTrianglesThreads.blocks, veinTrianglesThreads.threadsPerBlock);
				HANDLE_ERROR(hipPeekAtLastError());

				if constexpr (useBloodFlow)
				{
					HandleVeinEnd(bloodCells, streams);
					HANDLE_ERROR(hipPeekAtLastError());
				}

			}, particleGrid, triangleGrid);
	}
}