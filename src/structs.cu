#include "hip/hip_runtime.h"
#include "structs.cuh"
#include "defines.cuh"

cudaVec3::cudaVec3(int n)
{
	size = n;
	// allocate
	HANDLE_ERROR(hipMalloc((void**)&x, n * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&y, n * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&z, n * sizeof(float)));
}

__device__ float3 cudaVec3::get(int index)
{
	return make_float3(x[index], y[index], z[index]);
}

__device__ void cudaVec3::set(int index, float3 v)
{
	x[index] = v.x;
	y[index] = v.y;
	z[index] = v.z;
}

__device__ void cudaVec3::add(int index, float3 v)
{
	x[index] += v.x;
	y[index] += v.y;
	z[index] += v.z;
}