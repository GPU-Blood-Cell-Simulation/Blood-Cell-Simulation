#include "hip/hip_runtime.h"
#include "simulation.cuh"
#include "defines.cuh"
#include "physics.cuh"

#include <cmath>
#include <ctime>

#include "hip/hip_runtime.h"
#include ""


namespace sim
{

	__global__ void setupCurandStatesKernel(hiprandState* states, const unsigned long seed, const int particleCount);

	__global__ void generateRandomPositionsKernel(hiprandState* states, Particles particles, const int particleCount);


	// Allocate GPU buffers for the position vectors
	void allocateMemory(UniformGrid& grid, const unsigned int particleCount)
	{
		HANDLE_ERROR(hipMalloc((void**)&grid.cellIds, particleCount * sizeof(unsigned int)));
		HANDLE_ERROR(hipMalloc((void**)&grid.particleIds, particleCount * sizeof(unsigned int)));

		HANDLE_ERROR(hipMalloc((void**)&grid.cellStarts, width / cellWidth * height / cellHeight * depth / cellDepth * sizeof(unsigned int)));
		HANDLE_ERROR(hipMalloc((void**)&grid.cellEnds, width / cellWidth * height / cellHeight * depth / cellDepth * sizeof(unsigned int)));
	}


	// Generate initial positions and velocities of particles
	void generateRandomPositions(Particles particles, const int particleCount)
	{
		int threadsPerBlock = particleCount > 1024 ? 1024 : particleCount;
		int blocks = (particleCount + threadsPerBlock - 1) / threadsPerBlock;

		// Set up random seeds
		hiprandState* devStates;
		hipMalloc(&devStates, particleCount * sizeof(hiprandState));
		srand(static_cast<unsigned int>(time(0)));
		int seed = rand();
		setupCurandStatesKernel << <blocks, threadsPerBlock >> > (devStates, seed, particleCount);

		// Generate random positions and velocity vectors

		generateRandomPositionsKernel << <blocks, threadsPerBlock >> > (devStates, particles, particleCount);

		hipFree(devStates);
	}

	__global__ void setupCurandStatesKernel(hiprandState* states, const unsigned long seed, const int particleCount)
	{
		int id = blockIdx.x * blockDim.x + threadIdx.x;
		if (id >= particleCount)
			return;
		hiprand_init(seed, id, 0, &states[id]);
	}

	// Generate random positions and velocities at the beginning
	__global__ void generateRandomPositionsKernel(hiprandState* states, Particles p, const int particleCount)
	{
		int id = blockIdx.x * blockDim.x + threadIdx.x;
		if (id >= particleCount)
			return;

		p.position.x[id] = hiprand_uniform(&states[id]) * width / 3 + width/6;
		p.position.y[id] = hiprand_uniform(&states[id]) * height / 3 + height/6;
		p.position.z[id] = hiprand_uniform(&states[id]) * depth / 3 + depth/6;

		p.force.x[id] = 0;
		p.force.y[id] = 0;
		p.force.z[id] = 0;
	}


	__global__ void detectCollisions(BloodCells cells, unsigned int* cellIds, unsigned int* particleIds,
		unsigned int* cellStarts, unsigned int* cellEnds)
	{
		int id = blockIdx.x * blockDim.x + threadIdx.x;
		if (id >= cells.particlesCnt)
			return;

		int particleId = particleIds[id];
		float3 p1 = cells.particles.position.get(particleId);


		// Naive implementation
		/*for (int i = 0; i < particleCount; i++)
		{
			if (id == i || i == secondParticle)
				continue;

			float3 p2 = particles.position.get(i);
			if (length(p1 - p2) <= 5.0f)
			{
				particles.force.set(id, 50.0f * normalize(p1 - p2));
			}
		}*/

		// Using uniform grid

		int cellId = cellIds[id];

		for (int i = cellStarts[cellId]; i <= cellEnds[cellId]; i++)
		{
			int secondParticleId = particleIds[i];
			if (particleId == secondParticleId)
				continue;

			float3 p2 = cells.particles.position.get(secondParticleId);
			if (length(p1 - p2) <= 5.0f)
			{
				// Uncoalesced writes - area for optimization
				cells.particles.force.set(particleId, 50.0f * normalize(p1 - p2));
			}
		}
	}

	void calculateNextFrame(BloodCells cells, DeviceTriangles triangles, UniformGrid& grid, unsigned int trianglesCount)
	{
		// 1. calculate grid
		grid.calculateGrid(cells.particles);

		int threadsPerBlock = cells.particlesCnt > 1024 ? 1024 : cells.particlesCnt;
		int blDim = std::ceil(float(cells.particlesCnt) / threadsPerBlock);
		
		// 2. TODO: detections

		detectCollisions << < dim3(blDim), threadsPerBlock >> > (cells, grid.cellIds, grid.particleIds,
			grid.cellStarts, grid.cellEnds);


		physics::propagateParticles << < dim3(blDim), threadsPerBlock >> > (cells, triangles, trianglesCount);
		cells.PropagateForces();
	}
}