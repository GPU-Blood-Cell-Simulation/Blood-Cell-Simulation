#include "objects.cuh"

#include "hip/hip_runtime.h"
#include ""

// cudaVec3
void cudaVec3::createVec(int n)
{
    size = n;
	// allocate 
	HANDLE_ERROR(hipMalloc((void**)&x, n * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&y, n * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&z, n * sizeof(float)));
}

void cudaVec3::freeVec()
{
	hipFree(x);
	hipFree(y);
	hipFree(z);
}
