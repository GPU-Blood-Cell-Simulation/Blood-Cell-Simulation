#include "hip/hip_runtime.h"
#include "objects.cuh"
#include "defines.cuh"
#include "utilities.cuh"

// cudaVec3
void cudaVec3::createVec(int n)
{
    size = n;
	// allocate 
	HANDLE_ERROR(hipMalloc((void**)&x, n * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&y, n * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&z, n * sizeof(float)));
}

__device__ float3 cudaVec3::get(int index)
{
	return make_float3(x[index], y[index], z[index]);
}

__device__ void cudaVec3::set(int index, float3 v)
{
	x[index] = v.x;
	y[index] = v.y;
	z[index] = v.z;
}

__device__ void cudaVec3::add(int index, float3 v)
{
	x[index] += v.x;
	y[index] += v.y;
	z[index] += v.z;
}

// corpuscles

corpuscles::corpuscles(int n)
{
    centers.createVec(n);
}

// dipols 

__device__ void dipols::propagateForces(particles& gp, int particleInd)
{
    int secondParticle = particleInd%2 == 0 ? particleInd + 1 : particleInd - 1;
	float3 p1 = gp.position.get(particleInd);
	float3 p2 = gp.position.get(secondParticle);
	float3 v1 = gp.velocity.get(particleInd);
	float3 v2 = gp.velocity.get(secondParticle);
    
	float Fr = (length(p1 - p2) - L0) * k_sniff + dot(normalize(p1 - p2), (v1 - v2)) * d_fact;
    
	gp.force.add(particleInd, Fr * normalize(p2 - p1));
	gp.force.add(secondParticle, Fr * normalize(p1 - p2));
}

__device__ void dipols::setCorpuscle(int index, float3 center, particles& particls, int p_cnt)
{
	if(2*index < p_cnt)
	{
		centers.set(index, center);
		particls.position.set(2 * index,	 make_float3(0, 0, -L0) + center);
		particls.position.set(2 * index + 1, make_float3(0, 0,  L0) + center);

		particls.velocity.set(2 * index,	 make_float3(0, 0, v0));
		particls.velocity.set(2 * index + 1, make_float3(0, 0, v0));

		particls.force.set(2 * index,	  make_float3(0, 0, 0));
		particls.force.set(2 * index + 1, make_float3(0, 0, 0));
	}
}