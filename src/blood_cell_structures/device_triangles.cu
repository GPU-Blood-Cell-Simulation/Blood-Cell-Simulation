#include "hip/hip_runtime.h"
#include "device_triangles.cuh"

#include "../utilities/cuda_handle_error.cuh"
#include "../utilities/math.cuh"

#include "hip/hip_runtime.h"
#include ""
#include <algorithm>


__global__ void calculateCenters(cudaVec3 position, unsigned int* indices, cudaVec3 centers, int triangleCount)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id >= triangleCount)
		return;
	float3 vv1 = position.get(indices[3 * id]);
	float3 vv2 = position.get(indices[3 * id + 1]);
	float3 vv3 = position.get(indices[3 * id + 2]);

	float x = (vv1.x + vv2.x + vv3.x) / 3;
	float y = (vv1.y + vv2.y + vv3.y) / 3;
	float z = (vv1.z + vv2.z + vv3.z) / 3;
	centers.set(id, make_float3(x, y, z));
}

DeviceTriangles::DeviceTriangles(const Mesh& mesh) : triangleCount(mesh.indices.size() / 3), vertexCount(mesh.vertices.size()),
	centers(triangleCount), position(vertexCount), velocity(vertexCount), force(vertexCount), tempForceBuffer(vertexCount),
	threadsPerBlock(vertexCount > 1024 ? 1024 : vertexCount), blDim(std::ceil(float(vertexCount) / threadsPerBlock))
{
	// allocate
	HANDLE_ERROR(hipMalloc((void**)&indices, 3 * triangleCount * sizeof(int)));

	std::vector<unsigned int> indicesMem = mesh.indices;

	std::vector<float> vx(vertexCount);
	std::vector<float> vy(vertexCount);
	std::vector<float> vz(vertexCount);

	int iter = 0;
	std::for_each(mesh.vertices.begin(), mesh.vertices.end(), [&](auto& v)
		{
		vx[iter] = v.Position.x;
		vy[iter] = v.Position.y;
		vz[iter++] = v.Position.z;
		});

	// copy
	HANDLE_ERROR(hipMemcpy(indices, indicesMem.data(), 3 * triangleCount * sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(position.x, vx.data(), vertexCount * sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(position.y, vy.data(), vertexCount * sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(position.z, vz.data(), vertexCount * sizeof(float), hipMemcpyHostToDevice));

	// centers
	int threadsPerBlock = triangleCount > 1024 ? 1024 : triangleCount;
	int blocks = (triangleCount + threadsPerBlock - 1) / threadsPerBlock;
	calculateCenters << <blocks, threadsPerBlock >> > (position, indices, centers, triangleCount);
	hipDeviceSynchronize();
}

DeviceTriangles::DeviceTriangles(const DeviceTriangles& other) : isCopy(true), triangleCount(other.triangleCount), vertexCount(other.vertexCount),
	position(other.position), velocity(other.velocity), force(other.force), indices(other.indices), centers(other.centers), tempForceBuffer(other.tempForceBuffer),
	blDim(other.blDim), threadsPerBlock(other.threadsPerBlock) {}

DeviceTriangles::~DeviceTriangles()
{
	if (!isCopy)
	{
		HANDLE_ERROR(hipFree(indices));
	}
}

__global__ void propagateForcesIntoPositionsKernel(DeviceTriangles triangles)
{
	int vertex = blockDim.x * blockIdx.x + threadIdx.x;

	if (vertex >= triangles.vertexCount)
		return;

	// propagate forces into velocities
	triangles.velocity.add(vertex, dt * triangles.force.get(vertex));

	// propagate velocities into positions
	triangles.position.add(vertex, dt * triangles.velocity.get(vertex));

	// zero forces
	triangles.force.set(vertex, make_float3(0, 0, 0));
}

/// <summary>
/// Propagate forces -> velocities and velocities->positions
/// </summary>
void DeviceTriangles::propagateForcesIntoPositions()
{
	propagateForcesIntoPositionsKernel << <blDim, threadsPerBlock >> > (*this);
}


/// <summary>
/// Update the tempForceBuffer based on forces applied onto 4 neighboring vertices in 2D space uisng elastic springs
/// </summary>
/// <param name="force">Vertex force vector</param>
/// <param name="tempForceBuffer">Temporary buffer necessary to synchronize</param>
/// <returns></returns>
__global__ void gatherForcesKernel(DeviceTriangles triangles)
{
	// TODO: vertex distances (spring lengths) are hardcoded for now, ideally we'd like to calculate them for every possible vein model
	int vertex = blockDim.x * blockIdx.x + threadIdx.x;
	if (vertex >= triangles.force.size)
		return;

	float springForce;
	float3 neighborPosition;

	float3 vertexPosition = triangles.position.get(vertex);
	float3 vertexVelocity = triangles.position.get(vertex);
	float3 vertexForce = { 0,0,0 };

	// Calculate our own spatial indices
	unsigned int i = vertex / horizontalLayers;
	unsigned int j = vertex - i * horizontalLayers;

	// vertically adjacent vertices

	unsigned int jPrev = j != 0 ? j - 1 : horizontalLayers - 1;
	unsigned int jNext = (j + 1) % horizontalLayers;
	unsigned int vertexHorizontalPrev = i * horizontalLayers + jPrev;
	unsigned int vertexHorizontalNext = i * horizontalLayers + jNext;


	// Previous horizontally
	neighborPosition = triangles.position.get(vertexHorizontalPrev);
	springForce = triangles.calculateVeinSpringForce(vertexPosition, neighborPosition, vertexVelocity, triangles.velocity.get(vertexHorizontalPrev), 20.9057f, i, j, jPrev);
	vertexForce = vertexForce + springForce * normalize(neighborPosition - vertexPosition);

	// Next horizontally
	neighborPosition = triangles.position.get(vertexHorizontalNext);
	springForce = triangles.calculateVeinSpringForce(vertexPosition, neighborPosition, vertexVelocity, triangles.velocity.get(vertexHorizontalNext), 20.9057f, i, j, jNext);
	vertexForce = vertexForce + springForce * normalize(neighborPosition - vertexPosition);

	// not the lower end of the vein
	if (i != 0)
	{
		// Previous vertically
		unsigned int vertexVerticalPrev = (i - 1) * horizontalLayers + j;
		neighborPosition = triangles.position.get(vertexVerticalPrev);
		springForce = triangles.calculateVeinSpringForce(vertexPosition, neighborPosition, vertexVelocity, triangles.velocity.get(vertexVerticalPrev), 5.26999f,i,j,i-1);
		vertexForce = vertexForce + springForce * normalize(neighborPosition - vertexPosition);
	}

	//// not the upper end of the vein
	if (i != verticalLayers - 1)
	{
		//Next vertically
		unsigned int vertexVerticalNext = (i + 1) * horizontalLayers + j;
		neighborPosition = triangles.position.get(vertexVerticalNext);
		springForce = triangles.calculateVeinSpringForce(vertexPosition, neighborPosition, vertexVelocity, triangles.velocity.get(vertexVerticalNext), 5.26999f,i,j,i+1);
		vertexForce = vertexForce + springForce * normalize(neighborPosition - vertexPosition);
	}

	triangles.tempForceBuffer.set(vertex, vertexForce);
}

__global__ void updateForcesKernel(cudaVec3 force, const cudaVec3 tempForceBuffer)
{
	int vertex = blockDim.x * blockIdx.x + threadIdx.x;
	if (vertex >= force.size)
		return;

	force.add(vertex, tempForceBuffer.get(vertex));
}

/// <summary>
/// Gather forces from neighboring vertices, synchronize and then update forces for each vertex
/// </summary>
void DeviceTriangles::gatherForcesFromNeighbors()
{
	gatherForcesKernel << <blDim, threadsPerBlock >> > (*this);

	// Global synchronize - unfortunately necessary as neighboring vertices are not limited to blocks

	updateForcesKernel << <blDim, threadsPerBlock >> > (force, tempForceBuffer);
}
