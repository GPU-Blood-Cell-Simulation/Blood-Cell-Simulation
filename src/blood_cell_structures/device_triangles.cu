#include "hip/hip_runtime.h"
#include "device_triangles.cuh"

#include "../utilities/cuda_handle_error.cuh"

#include "hip/hip_runtime.h"
#include ""
#include <algorithm>


__global__ void calculateCenters(cudaVec3 position, unsigned int* indices, cudaVec3 centers, int triangleCount)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id >= triangleCount)
		return;
	float3 vv1 = position.get(indices[3 * id]);
	float3 vv2 = position.get(indices[3 * id + 1]);
	float3 vv3 = position.get(indices[3 * id + 2]);

	float x = (vv1.x + vv2.x + vv3.x) / 3;
	float y = (vv1.y + vv2.y + vv3.y) / 3;
	float z = (vv1.z + vv2.z + vv3.z) / 3;
	centers.set(id, make_float3(x, y, z));
}

DeviceTriangles::DeviceTriangles(const Mesh& mesh) : triangleCount(mesh.indices.size() / 3), vertexCount(mesh.vertices.size()),
	centers(triangleCount), position(vertexCount), velocity(vertexCount), force(vertexCount)
{
	// allocate
	HANDLE_ERROR(hipMalloc((void**)&indices, 3 * triangleCount * sizeof(int)));

	std::vector<unsigned int> indicesMem = mesh.indices;

	std::vector<float> vx(vertexCount);
	std::vector<float> vy(vertexCount);
	std::vector<float> vz(vertexCount);

	int iter = 0;
	std::for_each(mesh.vertices.begin(), mesh.vertices.end(), [&](auto& v)
		{
		vx[iter] = v.Position.x;
		vy[iter] = v.Position.y;
		vz[iter++] = v.Position.z;
		});

	// copy
	HANDLE_ERROR(hipMemcpy(indices, indicesMem.data(), 3 * triangleCount * sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(position.x, vx.data(), vertexCount * sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(position.y, vy.data(), vertexCount * sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(position.z, vz.data(), vertexCount * sizeof(float), hipMemcpyHostToDevice));

	// centers
	int threadsPerBlock = triangleCount > 1024 ? 1024 : triangleCount;
	int blocks = (triangleCount + threadsPerBlock - 1) / threadsPerBlock;
	calculateCenters << <blocks, threadsPerBlock >> > (position, indices, centers, triangleCount);
	hipDeviceSynchronize();
}

DeviceTriangles::DeviceTriangles(const DeviceTriangles& other) : isCopy(true), triangleCount(other.triangleCount), vertexCount(other.vertexCount),
position(other.position), velocity(other.velocity), force(other.force), indices(other.indices), centers(other.centers) {}

DeviceTriangles::~DeviceTriangles()
{
	if (!isCopy)
	{
		HANDLE_ERROR(hipFree(indices));
	}
}
