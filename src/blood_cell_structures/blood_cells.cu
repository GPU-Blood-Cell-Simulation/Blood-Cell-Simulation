#include "hip/hip_runtime.h"
#include "blood_cells.cuh"
#include "../utilities/cuda_handle_error.cuh"
#include "../utilities/math.cuh"
#include "../defines.hpp"

#include <vector>

#include "hip/hip_runtime.h"


BloodCells::BloodCells(int cellCount, int particlesInCell, const float* graphDesc) :
	particles(cellCount * particlesInCell), particlesInCell(particlesInCell), particleCount(cellCount * particlesInCell)
{
	int graphSize = particlesInCell * particlesInCell;

	HANDLE_ERROR(hipMalloc(&springsGraph, sizeof(float) * graphSize));
	HANDLE_ERROR(hipMemcpy(springsGraph, graphDesc, sizeof(float) * graphSize, hipMemcpyHostToDevice));
}

BloodCells::BloodCells(const BloodCells& other) : isCopy(true), particles(other.particles), particlesInCell(other.particlesInCell),
particleCount(other.particleCount), springsGraph(other.springsGraph) {}


BloodCells::~BloodCells()
{
	if (!isCopy)
	{
		HANDLE_ERROR(hipFree(springsGraph));
	}
}


__global__ void PropagateForcesOnDevice(BloodCells cells);


void BloodCells::propagateForces()
{
	int threadsPerBlock = particleCount > 1024 ? 1024 : particleCount;
	int blocks = (particleCount + threadsPerBlock - 1) / threadsPerBlock;

	PropagateForcesOnDevice << <blocks, threadsPerBlock >> > (*this);
}


// TODO : split into two kernels with a temporary buffer - necessary synchronization
__global__ void PropagateForcesOnDevice(BloodCells cells)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int inCellIndex = index % cells.particlesInCell;

	if (index >= cells.particleCount)
		return;

	float3 pos = cells.particles.position.get(index);
	float3 velo = cells.particles.velocity.get(index);

	for (int neighbourCellindex = 0; neighbourCellindex < cells.particlesInCell; neighbourCellindex++)
	{
		float springLen = cells.springsGraph[inCellIndex * cells.particlesInCell + neighbourCellindex];

		if (springLen == NO_SPRING)
			continue;

		int neighbourIndex = index - inCellIndex + neighbourCellindex;

		float3 neighbourPos = cells.particles.position.get(neighbourIndex);
		float3 neighbourVelo = cells.particles.velocity.get(neighbourIndex);

		float springForce = cells.calculateParticleSpringForce(pos, neighbourPos, velo, neighbourVelo, springLen);

		cells.particles.force.add(index, springForce * normalize(neighbourPos - pos));
	}
}