﻿#include "hip/hip_runtime.h"
#include ""

#include "simulation.cuh"
#include "defines.cuh"
#include "objects.cuh"
#include "window.h"

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

//// NVIDIA GPU selector for devices with multiple GPUs (e.g. laptops)
//extern "C"
//{
//    __declspec(dllexport) unsigned long NvOptimusEnablement = 0x00000001;
//}

int main()
{
    unsigned int* cellIds = 0;
    unsigned int* particleIds = 0;
    unsigned int* cellStarts = 0;
    unsigned int* cellEnds = 0;

    // Choose which GPU to run on, change this on a multi-GPU system.
    HANDLE_ERROR(hipSetDevice(0));

    Window window(windowWidth, windowHeight);

    // Allocate memory
    particles particls(PARTICLE_COUNT);
    dipols corpscls = dipols(10);

    sim::allocateMemory(&cellIds, &particleIds, &cellStarts, &cellEnds, PARTICLE_COUNT);

    // Generate random positions
    sim::generateRandomPositions(particls, PARTICLE_COUNT);
    //sim::generateInitialPositionsInLayers(particls, corpscls, PARTICLE_COUNT, 3);

    // MAIN LOOP HERE - probably dictated by glfw

    while (!window.shouldClose())
    {
        window.clear();

        // Calculate particle positions using CUDA
        sim::calculateNextFrame(particls, corpscls, cellIds, particleIds, cellStarts, cellEnds, PARTICLE_COUNT);

        window.updateParticles(particls);
        window.calculateFPS();

        window.handleEvents();
    }

    // Cleanup
    hipFree(particls.position.x);
    hipFree(particls.position.y);
    hipFree(particls.position.z);
    hipFree(particls.velocity.x);
    hipFree(particls.velocity.y);
    hipFree(particls.velocity.z);
    hipFree(particls.force.x);
    hipFree(particls.force.y);
    hipFree(particls.force.z);
    hipFree(cellIds);
    hipFree(particleIds);
    hipFree(cellStarts);
    hipFree(cellEnds);

    HANDLE_ERROR(hipDeviceReset());

    return 0;
}