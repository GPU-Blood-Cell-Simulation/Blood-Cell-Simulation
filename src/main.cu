﻿#include "hip/hip_runtime.h"
#include ""

#include "simulation.cuh"
#include "defines.cuh"
#include "objects.cuh"
#include "graphics/glcontroller.cuh"

#include <glad/glad.h>
#include <GLFW/glfw3.h>
#include <sstream>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

//// NVIDIA GPU selector for devices with multiple GPUs (e.g. laptops)
//extern "C"
//{
//    __declspec(dllexport) unsigned long NvOptimusEnablement = 0x00000001;
//}

int main()
{
    float* positionX = 0;
    float* positionY = 0;
    float* positionZ = 0;
    unsigned int* cellIds = 0;
    unsigned int* particleIds = 0;
    unsigned int* cellStarts = 0;
    unsigned int* cellEnds = 0;

    // Choose which GPU to run on, change this on a multi-GPU system.
    HANDLE_ERROR(hipSetDevice(0));

    // OpenGL setup
#pragma region OpenGLsetup
    GLFWwindow* window;

    if (!glfwInit())
        return -1;
    glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3);
    glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 3);
    glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);

    // Create a windowed mode window and its OpenGL context
    window = glfwCreateWindow(windowWidth, windowHeight, "Blood Cell Simualtion", NULL, NULL);
    if (!window)
    {
        glfwTerminate();
        return -1;
    }

    // Make the window's context current
    glfwMakeContextCurrent(window);

    // Load GL and set the viewport to match window size
    gladLoadGL();
    glViewport(0, 0, windowWidth, windowHeight);

    glEnable(GL_DEPTH_TEST);
    glEnable(GL_CULL_FACE);

    // debug
    glEnable(GL_DEBUG_OUTPUT);

    double lastTime = glfwGetTime();
    int frameCount = 0;
#pragma endregion

    // Create a graphics controller
    graphics::GLController glController;

    // Allocate memory
    particles particls(PARTICLE_COUNT);
    dipols corpscls(PARTICLE_COUNT /2);

    sim::allocateMemory(&cellIds, &particleIds, &cellStarts, &cellEnds, PARTICLE_COUNT);

    // Generate random positions
    //sim::generateRandomPositions(positionX, positionY, positionZ, particleCount);
    sim::generateInitialPositionsInLayers(particls, corpscls, PARTICLE_COUNT, 3);

    // MAIN LOOP HERE - probably dictaded by glfw

    while (!glfwWindowShouldClose(window))
    {
        // Clear 
        glClearColor(1.00f, 0.75f, 0.80f, 1.0f);
        glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

        // Calculate particle positions using CUDA
        //sim::calculateNextFrame(positionX, positionY, positionZ, cellIds, particleIds, cellStarts, cellEnds, particleCount);

        // Pass positions to OpenGL
        glController.calculateOffsets(positionX, positionY, positionZ, PARTICLE_COUNT);

        // OpenGL render
#pragma region rendering
        
        glController.draw();
        glfwSwapBuffers(window);

        // Show FPS in the title bar
        double currentTime = glfwGetTime();
        double delta = currentTime - lastTime;
        if (delta >= 1.0)
        {
            double fps = double(frameCount) / delta;
            std::stringstream ss;
            ss << "Blood Cell Simulation" << " " << " [" << fps << " FPS]";

            glfwSetWindowTitle(window, ss.str().c_str());
            lastTime = currentTime;
            frameCount = 0;
        }
        else
        {
            frameCount++;
        }

        glfwPollEvents();
#pragma endregion
    }

    // Cleanup
    hipFree(positionX);
    hipFree(positionY);
    hipFree(positionZ);
    hipFree(cellIds);
    hipFree(particleIds);
    hipFree(cellStarts);
    hipFree(cellEnds);

    glfwTerminate();

    HANDLE_ERROR(hipDeviceReset());

    return 0;
}